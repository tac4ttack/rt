#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>
#include <stdbool.h>
#include <stdint.h>
#include "ft_maths.hu"


static void HandleError(hipError_t err, const char *file,	int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		hipDeviceReset();
		// exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))


#define BACKCOLOR 0x00999999

#define EPSILON 0.00000000000000000000005
#define EPSILONF 0.0000000005F
#define MAX_DIST 10000000.0
#define SHADOW_BIAS 1000

#define CAM scene->cameras
#define CONES scene->cones
#define CYLIND scene->cylinders
#define LIGHT scene->lights
#define PLANE scene->planes
#define SPHERE scene->spheres
#define ACTIVECAM scene->cameras[scene->active_cam]

#define OPTION_WAVE 				(1 << 1)
#define OPTION_SEPIA				(1 << 2)
#define OPTION_BW					(1 << 3)
#define OPTION_RUN					(1 << 4)
#define OPTION_INVERT				(1 << 7)
#define OPTION_CARTOON_FOUR			(1 << 8)
#define OPTION_STEREO				(1 << 9)
#define OPTION_CARTOON_TWO			(1 << 10)
#define OPTION_SKYBOX				(1 << 11)

#define OBJ_FLAG_WAVES				(1 << 1)
#define OBJ_FLAG_CHECKERED			(1 << 2)
#define OBJ_FLAG_DIFF_MAP			(1 << 3)
#define OBJ_FLAG_BUMP_MAP			(1 << 4)
#define OBJ_FLAG_PLANE_LIMIT		(1 << 5)
#define OBJ_FLAG_PLANE_LIMIT_FIX	(1 << 6)
#define OBJ_FLAG_CUT				(1 << 7)

# define OBJ_CAM					1
# define OBJ_LIGHT					2
# define OBJ_CONE					3
# define OBJ_CYLINDER				4
# define OBJ_PLANE					5
# define OBJ_SPHERE					6
# define OBJ_ELLIPSOID				7
# define OBJ_THOR					8
# define OBJ_KUBE					9

typedef struct			s_gen
{
	size_t				mem_size;
	size_t				length;
	void				*mem;
	bool				(*add)(struct s_gen *, void *);
	bool				(*remove_mem_index)(struct s_gen *, size_t);
	bool				(*remove_index)(struct s_gen *, size_t);
	void				(*print)(struct s_gen *, \
								void (*)(struct s_gen *, void *));
}						t_gen;

/*
** CAM AND LIGHT STRUCTS ///////////////////////////////////////////////////////
*/


typedef struct			s_cam
{
	unsigned int		id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float				fov;
	float				pitch;
	float				yaw;
	float				roll;
}						t_cam;

typedef struct			s_light
{
	int					size;
	int					type;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	int					shrink;
	float				brightness;
	int					color;
}						t_light;

typedef struct			s_object
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;
}						t_object;

typedef struct			s_cone
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;

	float				angle;
	float3				u_axis;
	float				_align13;
}						t_cone;

typedef struct			s_cylinder
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;

	float3				base_dir;
	float				_align13;
	float				radius;
	float3				u_axis;
	float				_align14;
}						t_cylinder;

typedef struct			s_plane
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				normal;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;

	float				radius;
	float3				u_axis;
	float				_align13;
}						t_plane;

typedef struct			s_sphere
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;

	float				radius;
}						t_sphere;

typedef struct			s_ellipsoid
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;

	float				radius;
	float3				axis_size;
	float				_align13;
}						t_ellipsoid;

typedef struct			s_thor
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;

	double				lil_radius;
	double				big_radius;
}						t_thor;

typedef struct			s_kube
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float				_align0;
	float3				dir;
	float				_align1;
	float3				diff;
	float				_align2;
	float3				spec;
	float				_align3;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float				_align4;
	float3				limit_dir;
	float				_align5;
	float3				waves_p1;
	float				_align6;
	float3				waves_p2;
	float				_align7;
	float3				check_size;
	float				_align8;
	int					diff_map_id;
	float3				diff_offset;
	float				_align9;
	float3				diff_ratio;
	float				_align10;
	float3				cut_min;
	float				_align11;
	float3				cut_max;
	float				_align12;

	double				option;
}						t_kube;

////////////////////////////////////////////////////////////////////////////////

/*
** TREE OF RAYS STRUCT /////////////////////////////////////////////////////////
*/
typedef	struct			s_tor
{
	int					activate;
	float3				pos;
	float3				prim;
	float3				normale;
	float				coef_ref;
	float				coef_tra;
	float				opacity;
	unsigned int		color;
	float				ratio;
	int					type;

}						t_tor;
////////////////////////////////////////////////////////////////////////////////

/*
** CORE STRUCTS ////////////////////////////////////////////////////////////////
*/
typedef struct			s_light_ray
{
	float3				dir;
	float				dist;
}						t_light_ray;

typedef struct			s_hit
{
	float				dist;
	float3				normal;
	float3				pos;
	t_object 			*obj;
	int					mem_index;
	float				opacity;
	unsigned int		color;
	int					wall;
	int					lock;
}						t_hit;

typedef struct			s_ret
{
	int					wall;
	float				dist;
	float3				normal;
}						t_ret;

typedef struct			s_scene
{
	unsigned int		n_cams;
	unsigned int		active_cam;
	unsigned int		win_w;
	unsigned int		win_h;
	float3				ambient;
	float				_align0;
	unsigned int		mou_x;
	unsigned int		mou_y;
	unsigned int		depth;
	float				u_time;
	int					flag;
	unsigned int		over_sampling;
	unsigned int		mem_size_obj;
	unsigned int		mem_size_lights;
	t_cam				*cameras;
	void				*mem_lights;
	void				*mem_obj;
	unsigned int		*texture_earth;
	unsigned int		*texture_moon;
	unsigned int		*texture_earth_cloud;
	unsigned int		*texture_star;
}						t_scene;

typedef struct			s_cuda
{
	size_t				nb_mem;
	void				**mem;
	bool				(*add_buffer)(struct s_cuda *, size_t);
	bool				(*update_buffer)(struct s_cuda *, size_t);
}						t_cuda;

typedef struct			s_tex
{
	unsigned int		*pixel_array;
	int					width;
	int					height;
}						t_tex;


// OCL TO CUDA -> need tests
__device__ unsigned int	skybox(float3 dir, unsigned int *texture, int t_width, int t_height)
{
	unsigned int	color = 0;
	int2			uv = make_int2(0);

	uv.x = (int)floor((0.5 + (atan2(dir.z, dir.x) / (2 * M_PI))) * t_width);
	uv.y = (int)floor((0.5 - (asin(dir.y) / M_PI)) * t_height);
	if (uv.x < 0)
		uv.x = -uv.x;
	if (uv.y < 0)
		uv.y = -uv.y;
	if (uv.x >= t_width)
		uv.x %= (t_width - 1);
	if (uv.y >= t_height)
		uv.y %= (t_height - 1);
	color = texture[uv.x + uv.y * t_width];
	return (color);
}

// OCL TO CUDA -> OK
inline __device__ float radians(double degree) {
  return (degree * M_PI / 180.f);
}

// OCL TO CUDA -> need tests
__device__ float3	vector_get_rotate(float3 *me, float3 *rot)
{
	float3		n = make_float3(0.f);
	float		tmp = 0.f;

	n = *me;
	if (rot->x)
	{
		tmp = n.y * cosf(rot->x) - n.z * sinf(rot->x);
		n.z = n.y * sinf(rot->x) + n.z * cosf(rot->x);
		n.y = tmp;
	}
	if (rot->y)
	{
		tmp = n.x * cosf(rot->y) + n.z * sinf(rot->y);
		n.z = n.x * -sinf(rot->y) + n.z * cosf(rot->y);
		n.x = tmp;
	}
	if (rot->z)
	{
		tmp = n.x * cosf(rot->z) - n.y * sinf(rot->z);
		n.y = n.x * sinf(rot->z) + n.y * cosf(rot->z);
		n.x = tmp;
	}
	return (n);
}

// OCL TO CUDA -> need tests
__device__ float3	vector_get_inverse(float3 *me, float3 *rot)
{
	float3		n = make_float3(0.f);
	float		tmp = 0.f;

	n = *me;
	if (rot->z)
	{
		tmp = n.x * cosf(rot->z) - n.y * -sinf(rot->z);
		n.y = n.x * -sinf(rot->z) + n.y * cosf(rot->z);
		n.x = tmp;
	}
	if (rot->y)
	{
		tmp = n.x * cosf(rot->y) + n.z * -sinf(rot->y);
		n.z = n.x * sinf(rot->y) + n.z * cosf(rot->y);
		n.x = tmp;
	}
	if (rot->x)
	{
		tmp = n.y * cosf(rot->x) - n.z * -sinf(rot->x);
		n.z = n.y * -sinf(rot->x) + n.z * cosf(rot->x);
		n.y = tmp;
	}
	return (n);
}

// OCL TO CUDA -> OK
__device__ t_hit	hit_init(void)
{
	t_hit		hit;

	hit.dist = 0.f;
	hit.normal = make_float3(0.f);
	hit.obj = 0;
	hit.lock = 0;
	hit.wall = 0;
	hit.color = 0;
	hit.pos = make_float3(0.f);
	hit.mem_index = 0;
	hit.opacity = 0;
	return (hit);
}

// OCL TO CUDA -> need tests
__device__ float3	rotat_zyx(float3 vect, float pitch, float yaw, float roll)
{
	float3		res = make_float3(0.f);
	float		rad_pitch = radians(pitch);
	float		rad_yaw = radians(yaw);
	float		rad_roll = radians(roll);

	res.x = vect.x * cosf(rad_roll) * cosf(rad_yaw) + vect.y * (cosf(rad_pitch) * -sinf(rad_roll) + cosf(rad_roll) * sinf(rad_yaw) * sinf(rad_pitch)) + vect.z * (-sinf(rad_roll) * -sinf(rad_pitch) + cosf(rad_roll) * sinf(rad_yaw) * cosf(rad_pitch));
	res.y = vect.x * sinf(rad_roll) * cosf(rad_yaw) + vect.y * (cosf(rad_roll) * cosf(rad_pitch) + sinf(rad_roll) * sinf(rad_yaw) * sinf(rad_pitch)) + vect.z * (cosf(rad_roll) * -sinf(rad_pitch) + sinf(rad_roll) * sinf(rad_yaw) * cosf(rad_pitch));
	res.z = vect.x * -sinf(rad_yaw) + vect.y * cosf(rad_yaw) * sinf(rad_pitch) + vect.z * cosf(rad_yaw) * cosf(rad_pitch);
	return (res);
}

// OCL TO CUDA -> need tests
__device__ float3	rotat_xyz(float3 vect, float pitch, float yaw, float roll)
{
	float3		res = make_float3(0.f);
	float		rad_pitch = radians(pitch);
	float		rad_yaw = radians(yaw);
	float		rad_roll = radians(roll);

	res.x = vect.x * cos(rad_yaw) * cos(rad_roll) + vect.y * cos(rad_yaw) * -sin(rad_roll) + vect.z * sin(rad_yaw);
	res.y = vect.x * (-sin(rad_pitch) * -sin(rad_yaw) * cos(rad_roll) + cos(rad_pitch) * sin(rad_roll)) + vect.y * (-sin(rad_pitch) * -sin(rad_yaw) * -sin(rad_roll) + cos(rad_pitch) * cos(rad_roll)) + vect.z * cos(rad_yaw) * -sin(rad_pitch);
	res.z = vect.x * (cos(rad_pitch) * -sin(rad_yaw) * cos(rad_roll) + sin(rad_pitch) * sin(rad_roll)) + vect.y * (cos(rad_pitch) * -sin(rad_yaw) * -sin(rad_roll) + sin(rad_pitch) * cos(rad_roll)) + vect.z * cos(rad_yaw) * cos(rad_pitch);
	return (res);
}

// OCL TO CUDA -> need tests
__device__ float3	rotat_x(float3 vect, float angle)
{
	float3 		res = make_float3(0.f);
	float		teta = radians(angle);

	res.x = (vect.x * 1.f) + (vect.y * 0.f) + (vect.z * 0.f);
	res.y = (vect.x * 0.f) + (vect.y * cos(teta)) + (vect.z * -sin(teta));
	res.z = (vect.x * 0.f) + (vect.y * sin(teta)) + (vect.z * cos(teta));
	return (res);
}

// OCL TO CUDA -> need tests
__device__ float3	rotat_y(float3 vect, float angle)
{
	float3 		res = make_float3(0.f);
	float		teta = radians(angle);

	res.x = (vect.x * cos(teta)) + (vect.y * 0.f) + (vect.z * sin(teta));
	res.y = (vect.x * 0.f) + (vect.y * 1) + (vect.z * 0.f);
	res.z = (vect.x * -sin(teta)) + (vect.y * 0.f) + (vect.z * cos(teta));
	return (res);
}

// OCL TO CUDA -> need tests
__device__ float3	rotat_z(float3 vect, float angle)
{
	float3 		res = make_float3(0.f);
	float		teta = radians(angle);

	res.x = (vect.x * cos(teta)) + (vect.y * -sin(teta)) + (vect.z * 0.f);
	res.y = (vect.x * sin(teta)) + (vect.y * cos(teta)) + (vect.z * 0.f);
	res.z = (vect.x * 0.f) + (vect.y * 0.f) + (vect.z * 1.f);
	return (res);
}



// OCL TO CUDA -> need tests (is unused in kernel)
__device__ unsigned int	blend_multiply(unsigned int c1, unsigned int c2)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);
	unsigned int r2 = (c2 & 0x00FF0000) >> 16;
	unsigned int g2 = (c2 & 0x0000FF00) >> 8;
	unsigned int b2 = (c2 & 0x000000FF);

	r = (r1 * r2 > 255 ? 255 : r1 * r2);
	g = (g1 * g2 > 255 ? 255 : g1 * g2);
	b = (b1 * b2 > 255 ? 255 : b1 * b2);

	return ((r << 16) + (g << 8) + b);
}

// OCL TO CUDA -> need tests in use
__device__ unsigned int	blend_med(unsigned int c1, unsigned int c2)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);
	unsigned int r2 = (c2 & 0x00FF0000) >> 16;
	unsigned int g2 = (c2 & 0x0000FF00) >> 8;
	unsigned int b2 = (c2 & 0x000000FF);

	r = (r1 + r2) / 2;
	g = (g1 + g2) / 2;
	b = (b1 + b2) / 2;
	return ((r << 16) + (g << 8) + b);
}

// OCL TO CUDA -> OK
__device__ unsigned int	blend_add(unsigned int c1, unsigned int c2)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);
	unsigned int r2 = (c2 & 0x00FF0000) >> 16;
	unsigned int g2 = (c2 & 0x0000FF00) >> 8;
	unsigned int b2 = (c2 & 0x000000FF);

	r = (r1 + r2 > 255 ? 255 : r1 + r2);
	g = (g1 + g2 > 255 ? 255 : g1 + g2);
	b = (b1 + b2 > 255 ? 255 : b1 + b2);
	return ((r << 16) + (g << 8) + b);
}

// OCL TO CUDA -> OK
__device__ unsigned int	blend_factor(unsigned int c1, float factor)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);

	r = r1 * factor;
	g = g1 * factor;
	b = b1 * factor;
	return ((r << 16) + (g << 8) + b);
}

// OCL TO CUDA -> OK
__device__ unsigned int	get_ambient(t_scene *scene, unsigned int obj_color)
{
	unsigned int r, g, b;

	r = (obj_color & 0x00FF0000) >> 16;
	g = (obj_color & 0x0000FF00) >> 8;
	b = (obj_color & 0x000000FF);
	r = (0.01 + r * scene->ambient.x > 255 ? 255 : 0.01 + r * scene->ambient.x);
	g = (0.01 + g * scene->ambient.y > 255 ? 255 : 0.01 + g * scene->ambient.y);
	b = (0.01 + b * scene->ambient.z > 255 ? 255 : 0.01 + b * scene->ambient.z);
	return ((r << 16) + (g << 8) + b);
}

// OCL TO CUDA -> OK
__device__ unsigned int	sepiarize(unsigned int color)
{
	uint3	base, cooking_pot = make_uint3(0, 0, 0);
	base.x = (color & 0x00FF0000) >> 16;
	base.y = (color & 0x0000FF00) >> 8;
	base.z = (color & 0x000000FF);
	cooking_pot.x = (base.x * 0.393) + (base.y * 0.769) + (base.z * 0.189);
	cooking_pot.y = (base.x * 0.349) + (base.y * 0.686) + (base.z * 0.168);
	cooking_pot.z = (base.x * 0.272) + (base.y * 0.534) + (base.z * 0.131);
	(cooking_pot.x > 255 ? cooking_pot.x = 255 : 0);
	(cooking_pot.y > 255 ? cooking_pot.y = 255 : 0);
	(cooking_pot.z > 255 ? cooking_pot.z = 255 : 0);
	return (((uint)cooking_pot.x << 16) + ((uint)cooking_pot.y << 8) + (uint)cooking_pot.z);
}

// OCL TO CUDA -> OK
__device__ unsigned int	invert(unsigned int color)
{
	uint3	base = make_uint3(0, 0, 0);
	base.x = (color & 0x00FF0000) >> 16;
	base.y = (color & 0x0000FF00) >> 8;
	base.z = (color & 0x000000FF);
	base.x = 255 - base.x;
	base.y = 255 - base.y;
	base.z = 255 - base.z;
	return (((uint)base.x << 16) + ((uint)base.y << 8) + (uint)base.z);
}

// OCL TO CUDA -> OK
__device__ unsigned int	desaturate(unsigned int color)
{
	uint3	rgb = make_uint3(0, 0, 0);
	rgb.x = (color & 0x00FF0000) >> 16;
	rgb.y = (color & 0x0000FF00) >> 8;
	rgb.z = (color & 0x000000FF);
	float 	average = (rgb.x + rgb.y + rgb.z) / 3;
	return (((uint)average << 16) + ((uint)average << 8) + (uint)average);
}

// OCL TO CUDA -> OK
__device__ unsigned int cartoonize_four(unsigned int col_r, unsigned int col_g, unsigned int col_b)
{
		if (col_r > 0 && col_r <= 50)
			col_r = 32;
		else if (col_r > 50 && col_r <= 100)
			col_r = 96;
		else if (col_r > 100 && col_r <= 200)
			col_r = 160;
		else if (col_r > 200 && col_r < 250)
			col_r = 224;
		else if (col_r >= 255)
			col_r = 255;

		if (col_g > 0 && col_g <= 50)
			col_g = 32;
		else if (col_g > 50 && col_g <= 100)
			col_g = 96;
		else if (col_g > 100 && col_g <= 200)
			col_g = 160;
		else if (col_g > 200 && col_g < 250)
			col_g = 224;
		else if (col_g >= 255)
			col_g = 255;

		if (col_b > 0 && col_b <= 50)
			col_b = 32;
		else if (col_b > 50 && col_b <= 100)
			col_b = 96;
		else if (col_b > 100 && col_b <= 200)
			col_b = 160;
		else if (col_b > 200 && col_b < 250)
			col_b = 224;
		else if (col_b >= 255)
			col_b = 255;

	return (((col_r << 16) + (col_g << 8) + col_b));
}

// OCL TO CUDA -> OK
__device__ unsigned int cartoonize_two(unsigned int col_r, unsigned int col_g, unsigned int col_b)
{
		if (col_r > 0 && col_r <= 128)
			col_r = 64;
		else if (col_r > 128 && col_r <= 255)
			col_r = 192;
		else if (col_r >= 255)
			col_r = 255;

		if (col_g > 0 && col_g <= 128)
			col_g = 64;
		else if (col_g > 128 && col_g <= 255)
			col_g = 192;
		else if (col_g >= 255)
			col_g = 255;

		if (col_b > 0 && col_r <= 128)
			col_b = 64;
		else if (col_b > 128 && col_b <= 255)
			col_b = 192;
		else if (col_b >= 255)
			col_b = 255;

	return (((col_r << 16) + (col_g << 8) + col_b));
}

// OCL TO CUDA -> need tests
__device__ float	inter_plan_private(t_plane *plane, float3 ray, float3 origin)
{
	float		t = 0.f;

	t = dot(normalize(ray), plane->normal);
	if (fabs(t) < EPSILONF|| (plane->radius && t > plane->radius))
		return (0.f);
	t = (dot(plane->pos - origin, plane->normal)) / t;
	if (t < EPSILONF)
		return (0.f);
	return (t);
}

// OCL TO CUDA -> need tests
__device__ t_ret	object_limited(t_object *object,
							float res1, float res2,
							float3 ray, float3 origin)
{
	t_ret		ret;
	t_plane		t;
	float		dist_plan = 0.f;

	ret.dist = 0.f;
	ret.wall = 0.f;
	ret.normal = make_float3(0.f);
	t.pos = object->limit_pos;
	t.normal = object->limit_dir;
	t.radius = 0.f;
	dist_plan = inter_plan_private(&t, ray, origin);

	// IN OBJECT
	if (res1 < EPSILONF)
	{
		if (dot(t.normal, ray) > EPSILONF)
		{
			if (!dist_plan && res2 > EPSILONF)
				ret.dist = res2;
			else if (res2 < dist_plan)
				return (ret);
			else
			{
				ret.dist = dist_plan;
				ret.normal = -t.normal;
				ret.wall = 1;
			}
		}
		else
		{
			if (dist_plan > res2)
				ret.dist = res2;
			else if (dist_plan < MAX_DIST)
			{
				ret.dist = dist_plan;
				ret.normal = -t.normal;
				ret.wall = 1;
			}
		}
		return (ret);
	}

	if (dot(t.normal, ray) > EPSILONF)
	{
		if (res2 < dist_plan || dist_plan > MAX_DIST)
			return (ret);
		else if (res1 > dist_plan)
			ret.dist = res1;
		else if (dist_plan < MAX_DIST)
		{
			ret.dist = dist_plan;
			ret.normal = -t.normal;
			ret.wall = 1;
		}
	}
	else
	{
		if (dist_plan < res1)
			return (ret);
		ret.dist = res1;
	}
	return (ret);
}

// OCL TO CUDA -> need tests
__device__ unsigned int	plane_checkerboard(float3 normale, float3 pos, unsigned int color, float3 check_size)
{
	float3			u_axis = make_float3(0.f);
	float3			v_axis = make_float3(0.f);
	int2			uv = make_int2(0);

	u_axis.x = normale.y;
	u_axis.y = normale.z;
	u_axis.z = -normale.x;
	v_axis = cross(u_axis, normale);
	uv.x = (int)(floor(dot(pos, u_axis) / check_size.x));
	uv.y = (int)(floor(dot(pos, v_axis) / check_size.y));
	if (uv.x % 2 == 0)
	{
		if (uv.y % 2 == 0)
			return (0);
		else
			return (color);
	}
	else if (uv.y % 2 == 0)
		return (color);
	return (0);
}


// OCL TO CUDA -> need tests
__device__ unsigned int		plane_texture(float3 normale, float3 pos, float3 u_axis, float3 ratio, float3 offset, unsigned int *texture, int width, int height)
{
	float3			v_axis = make_float3(0.f);
	int2			uv = make_int2(0);

	v_axis = cross(u_axis, normale);
	uv.x = (int)(floor(dot(pos, u_axis) * ratio.x + offset.x));
	uv.y = (int)(floor(dot(pos, v_axis) * ratio.y + offset.y));
	uv.x %= width - 1;
	uv.y %= height - 1;
	if (uv.x < 0)
		uv.x = (uv.x + width - 1);
	if (uv.y < 0)
		uv.y = (uv.y + height - 1);
	uv.x %= width - 1;
	uv.y %= height - 1;
	return (texture[uv.y + uv.x * width]);
}

// OCL TO CUDA -> OK
__device__ bool		solve_quadratic(float a, float b, float c, float *inter0, float *inter1)
{
	float 		discr;
	float 		tmp;
	float 		q;

	q = 0;
	discr = b * b - 4.f * a * c;
	tmp = 0;
	if (discr < EPSILONF)
		return (false);
	else if (discr < EPSILONF)
	{
		*inter0 = -0.5 * b / a;
		*inter1 = *inter0;
	}
	else
	{
		q = (b > 0) ? (-0.5f * (b + sqrtf(discr))): (-0.5 * (b - sqrtf(discr)));
		*inter0 = q / a;
		*inter1 = c / q;
	}
	if (*inter0 > *inter1)
	{
		tmp = *inter0;
		*inter0 = *inter1;
		*inter1 = tmp;
	}
	if (*inter0 < 0)
	{
		if (*inter1 < 0)
			return (false);
	}
	return (true);
}

// OCL TO CUDA -> need tests in use 
__device__ double3	thor_get_rotate(double3 *that, float3 *rot)
{
	double3		n = make_double3(0.f);
	float		tmp = 0.f;

	n = *that;
	if (rot->x)
	{
		tmp = n.y * cos(rot->x) - n.z * sin(rot->x);
		n.z = n.y * sin(rot->x) + n.z * cos(rot->x);
		n.y = tmp;
	}
	if (rot->y)
	{
		tmp = n.x * cos(rot->y) + n.z * sin(rot->y);
		n.z = n.x * -sin(rot->y) + n.z * cos(rot->y);
		n.x = tmp;
	}
	if (rot->z)
	{
		tmp = n.x * cos(rot->z) - n.y * sin(rot->z);
		n.y = n.x * sin(rot->z) + n.y * cos(rot->z);
		n.x = tmp;
	}
	return (n);
}

// OCL TO CUDA -> need tests in use
__device__ double	ft_ret(double *tab)
{
	double		ret = -1.0f;
	int			i = 0;

	while(i < 4)
	{
		if(tab[i] > EPSILON && ret == -1)
			ret = tab[i];
			if (tab[i] < ret && tab[i] > EPSILON )
			ret = tab[i];
		i++;
	}
	if (ret == -1.0f)
		ret = 0.f;
	return (ret);
}

// OCL TO CUDA -> need tests in use
__device__ double3	ft_solve_3(double a, double b, double c, double d)
{
	double		a1 = 0.f;
	double		a2 = 0.f;
	double		a3 = 0.f;
	double3		Result = make_double3(0.f);
	double		theta = 0.f;
	double		sqrtQ = 0.f;
	double		e = 0.f;
	double		Q = 0.f;
	double		R = 0.f;
	double		Qcubed = 0.f;

	a1 = c / d;
	a2 = b / d;
	a3 = a / d;

	Q = (a1 * a1 - 3.0f * a2) / 9.0f;
	R = (2.0f * a1 * a1 * a1 - 9.0f * a1 * a2 + 27.0f * a3) / 54.0f;
	Qcubed = Q * Q * Q;
	d = Qcubed - R * R;
	if ( d >= EPSILON )
	{
		if ( Q < EPSILON )
		{
			Result.x = 0.f;
			Result.y = 0.f;
			Result.z = 0.f;
				return (Result);
		}
		theta = acos(R / sqrt(Qcubed));
		sqrtQ = sqrt(Q);
		Result.x = -2.0f * sqrtQ * cos(theta / 3.0f) - a1 / 3.0f;
		Result.y = -2.0f * sqrtQ * cos((theta + 2.0f * M_PI) / 3.0f ) - a1 / 3.0f;
		Result.z = -2.0f * sqrtQ * cos((theta + 4.0f * M_PI) / 3.0f ) - a1 / 3.0f;
	}
	else
	{
		e = pow(sqrt((double)-d) + fabs((double)R), (double)1.0f/ (double)3.0f);
		if ( R > EPSILON )
			e = -e;
		Result.x = Result.y = Result.z = (e + Q / e) - a1 / 3.0f;
	}
	return (Result);
}

// OCL TO CUDA -> need tests in use
__device__ double	ft_solve_4(double t[5])
{
	double		Result[4] = {0.f};
	double3		Roots = make_double3(0.f);
	double		Rsquare = 0;
	double		Rrec = 0;
	double		D = 0;
	double		E = 0;
	double3		b = make_double3(0.f);
	double		a0 = 0;
	double		a1 = 0;
	double		a2 = 0;
	double		a3 = 0;
	double		R = 0;
	double		y = 0;

	a0 = t[0] / t[4];
	a1 = t[1] / t[4];
	a2 = t[2] / t[4];
	a3 = t[3] / t[4];

	b.x = 4.0f * a2 * a0 - a1 * a1 - a3 * a3 * a0;
	b.y = a1 * a3 - 4.0f * a0;
	b.z = -a2;
	Roots = ft_solve_3(b.x, b.y, b.z, 1.0f);
	y = fmax(Roots.x, fmax(Roots.y, Roots.z));
	R = 0.25f * a3 * a3 - a2 + y;

	if (R < EPSILON)
		return (0.f);
	R = sqrt(R);
	if ( R == EPSILON )
	{
		D = sqrt( 0.75f * a3 * a3 - 2.0f * a2 + 2.0f * sqrt( y * y - 4.0f * a0 ) );
		E = sqrt( 0.75f * a3 * a3 - 2.0f * a2 - 2.0f * sqrt( y * y - 4.0f * a0 ) );
	}
	else
	{
		Rsquare = R * R;
		Rrec = 1.0f / R;
		D = sqrt( 0.75f * a3 * a3 - Rsquare - 2.0f * a2 + 0.25f * Rrec * (4.0f * a3 * a2 - 8.0f * a1 - a3 * a3 * a3) );
		E = sqrt( 0.75f * a3 * a3 - Rsquare - 2.0f * a2 - 0.25f * Rrec * (4.0f * a3 * a2 - 8.0f * a1 - a3 * a3 * a3) );
	}
	Result[0] = -0.25f * a3 + 0.5f * R + 0.5f * D;
	Result[1] = -0.25f * a3 + 0.5f * R - 0.5f * D;
	Result[2] = -0.25f * a3 - 0.5f * R + 0.5f * E;
	Result[3] = -0.25f * a3 - 0.5f * R - 0.5f * E;
	return(ft_ret(Result));
}


// OCL TO CUDA -> need tests in use 
__device__ t_ret		inter_thor(t_thor *thor, float3 ray, float3 origin)
{
	t_ret			ret;
	ret.dist = 0.f;
	ret.normal = make_float3(0.f);
	ret.wall = 0;

	double 		big_radius = thor->big_radius * thor->big_radius;
	double		lil_radius = thor->lil_radius * thor->lil_radius;

	double3		d_ray;
	d_ray.x = (double)ray.x;
	d_ray.y = (double)ray.y;
	d_ray.z = (double)ray.z;
	d_ray = thor_get_rotate(&d_ray, &thor->dir);

	double3		d_dir;
	d_dir.x = (double)origin.x - (double)thor->pos.x;
	d_dir.y = (double)origin.y - (double)thor->pos.y;
	d_dir.z = (double)origin.z - (double)thor->pos.z;
	d_dir = thor_get_rotate(&d_dir, &thor->dir);

	double3		k;
	k.x = (d_ray.x * d_ray.x) + (d_ray.y * d_ray.y) + (d_ray.z * d_ray.z);
	k.y = 4.0f * big_radius;
	k.z = (d_dir.x) * d_ray.x \
		+ (d_dir.y) * d_ray.y \
		+ (d_dir.z) * d_ray.z;

	double		e;
	e =	(d_dir.x) * (d_dir.x) + \
		(d_dir.y) * (d_dir.y) + \
		(d_dir.z) * (d_dir.z) - \
		big_radius - lil_radius;

	double		c[5];
	c[0] = e * e - k.y * (lil_radius - (d_dir.y) * (d_dir.y));
	c[1] = 4.0f * k.z * e + 2.0f * k.y * (d_dir.y) * d_ray.y;
	c[2] = 2.0f * k.x * e + 4.0f * k.z * k.z + k.y * d_ray.y * d_ray.y;
	c[3] = 4.0f * k.x * k.z;
	c[4] = k.x * k.x;

	ret.dist = ft_solve_4(c);
	return (ret);
}

// OCL TO CUDA -> need tests in use 
__device__ float3 get_thor_normal(t_thor *thor, float3 hitpos)
{
	float3	res = make_float3(0.f);
	float	c = 0.f;

	float	R = (float)((thor->lil_radius * thor->lil_radius));
	float	r = (float)((thor->big_radius * thor->big_radius));

	float3 pos = hitpos - thor->pos;
	pos = vector_get_rotate(&pos, &thor->dir);

	c = ((pos.x * pos.x) + (pos.y * pos.y) + (pos.z * pos.z) - r - R);

	res.x = 4.0f * c * pos.x;
	res.y = 4.0f * pos.y * (c + 2 * r);
	res.z = 4.0f * c * pos.z;

	res = vector_get_inverse(&res, &thor->dir);
	return (res);
}

__device__ t_ret		inter_kube(t_kube *kube, float3 ray, float3 origin)
{
	t_ret		ret;
	ret.dist = 0;
	ret.normal = make_float3(0.f);
	ret.wall = 0;

	double3		d_ray;
	d_ray.x = (double)ray.x;
	d_ray.y = (double)ray.y;
	d_ray.z = (double)ray.z;
	d_ray = thor_get_rotate(&d_ray, &kube->dir);

	double3		d_dir;
	d_dir.x = (double)origin.x - (double)kube->pos.x;
	d_dir.y = (double)origin.y - (double)kube->pos.y;
	d_dir.z = (double)origin.z - (double)kube->pos.z;
	d_dir = thor_get_rotate(&d_dir, &kube->dir);

	double		c[5];
	c[4] = (pow(d_ray.x, 4.0f) + pow(d_ray.y, 4.0f) + pow(d_ray.z, 4.0f));
	c[3] = 4.0f * ((pow(d_ray.x, 3.0f) * d_dir.x) + (pow(d_ray.y, 3.0f) * d_dir.y)+ (pow(d_ray.z, 3.0f) * d_dir.z));
	c[2] = 6.0f * ((pow(d_ray.x, 2.0f) * pow(d_dir.x, 2.0f) + pow(d_ray.y, 2.0f) * pow(d_dir.y, 2.0f) + pow(d_ray.z, 2.0f) * pow(d_dir.z, 2.0f))) - 5.0 * (pow(d_ray.x, 2.0f) + pow(d_ray.y, 2.0f) + pow(d_ray.z, 2.0f));
	c[1] = 4.0f * (pow(d_dir.x, 3.0f) * d_ray.x + pow(d_dir.y, 3.0f) * d_ray.y + pow(d_dir.z, 3.0f) * d_ray.z) - 10.0 * (d_dir.x * d_ray.x + d_dir.y * d_ray.y + d_dir.z * d_ray.z);
	c[0] = (pow(d_dir.x, 4.0f) + pow(d_dir.y, 4.0f) + pow(d_dir.z, 4.0f)) - 5.0 * (d_dir.x * d_dir.x + d_dir.y * d_dir.y +d_dir.z * d_dir.z) + kube->option;

	ret.dist = ft_solve_4(c);
	return (ret);
}

// KUBE
__device__ float3 get_kube_normal(t_kube *kube, float3 hitpos)
{
	float3 pos = hitpos - kube->pos;
	pos = vector_get_rotate(&pos, &kube->dir);
	float3  res = make_float3(0.f);

     res.x = 4.0f * powf(pos.x, 3.0f) - 10.0 * pos.x;
     res.y = 4.0f * powf(pos.y, 3.0f) - 10.0 * pos.y;
     res.z = 4.0f * powf(pos.z, 3.0f) - 10.0 * pos.z;

	 res = vector_get_inverse(&res, &kube->dir);
	 return (res);
}


// OCL TO CUDA -> need tests in use (earth texture missing)
__device__ unsigned int		sphere_texture(float3 pos, unsigned int *texture, int t_width, int t_height, float3 ratio, float3 offset)
{
	unsigned int	color = 0;
	int3			uv = make_int3(0);
	int3			size = make_int3(0);

	size.x = (int)(floor(t_width * ratio.x));
	size.y = (int)(floor(t_height * ratio.y));
	uv.x = (int)(floor((0.5 + (atan2(pos.z, pos.x) / (2 * M_PI))) * size.x + offset.x));
	uv.y = (int)(floor((0.5 - (asin(pos.y) / M_PI)) * size.y + offset.y));
	if (uv.x < 0)
	{
		uv.x %= t_width;
		uv.x = (uv.x - t_width) * -1;
	}
	else
		uv.x %= t_width;
	if (uv.y < 0)
		uv.y %= t_height;
	else
	{
		uv.y %= t_height;
		uv.y = (uv.y - t_height) * -1;
	}
	color = texture[uv.x + uv.y * t_width];
	return (color);
}

// OCL TO CUDA -> need tests
__device__ t_ret	sphere_cut(t_sphere *sphere, float3 ray, float3 origin, float res1, float res2)
{
	t_ret		ret;
	float3		pt_i1 = make_float3(0.f);
	float3		pt_i2 = make_float3(0.f);
	float3		bord1 = make_float3(0.f);
	float3		bord2 = make_float3(0.f);
	ret.dist = 0;
	ret.wall = 0;
	ret.normal = make_float3(0.f);

	bord1.x = sphere->pos.x + sphere->radius;    
	bord2.x = sphere->pos.x - sphere->radius;	  	
	bord1.y = sphere->pos.y + sphere->radius;		
	bord2.y = sphere->pos.y - sphere->radius;		
	bord1.z = sphere->pos.z + sphere->radius;		
	bord2.z = sphere->pos.z - sphere->radius;	

	pt_i1.x = origin.x + ray.x * res1;
	pt_i1.y = origin.y + ray.y * res1;
	pt_i1.z = origin.z + ray.z * res1;

	pt_i2.x = origin.x + ray.x * res2;
	pt_i2.y = origin.y + ray.y * res2;
	pt_i2.z = origin.z + ray.z * res2;
	
	if (pt_i1.x <= 	bord1.x - sphere->cut_max.x && pt_i1.y <= bord1.y - sphere->cut_max.y && pt_i1.z <= bord1.z - sphere->cut_max.z && pt_i1.x >= bord2.x + sphere->cut_min.x && pt_i1.y >= bord2.y + sphere->cut_min.y && pt_i1.z >= bord2.z + sphere->cut_min.z)
	{
		ret.dist = res1;
	}
	else if (pt_i2.x <= bord1.x - sphere->cut_max.x && pt_i2.y <= bord1.y - sphere->cut_max.y && pt_i2.z <= bord1.z - sphere->cut_max.z && pt_i2.x >= bord2.x + sphere->cut_min.x && pt_i2.y >= bord2.y + sphere->cut_min.y && pt_i2.z >= bord2.z + sphere->cut_min.z) 
	{
		ret.dist = res2;
	}
	return (ret);
}

// OCL TO CUDA -> OK
__device__ unsigned int		sphere_checkerboard(float3 dir, unsigned int color, float3 check_size)
{
	int2	uv = make_int2(0);

	uv.x = (int)(floor((0.5 + (atan2(dir.z, dir.x) / (2 * 3.1415))) * check_size.x));
	uv.y = (int)(floor((0.5 - (asin(dir.y) / 3.1415)) * check_size.y));
	if (uv.x % 2 == 0)
	{
		if (uv.y % 2 == 0)
			return (0);
		else
			return (color);
	}
	else if (uv.y % 2 == 0)
			return (color);
	return (0);
}

// OCL TO CUDA -> OK
__device__ float3	get_sphere_abc(float radius, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);

	abc.x = dot(ray, ray);
	abc.y = 2 * dot(ray, origin);
	abc.z = dot(origin, origin) - (radius * radius);
	return (abc);
}

__device__ t_ret	mini_inter_sphere(t_sphere *sphere, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);
	float		res1 = 0.f;
	float		res2 = 0.f;
	float3		pos = make_float3(0.f);
	t_ret		ret;

	ret.dist = 0.f;
	ret.wall = 0;
	ret.normal = make_float3(0.f);
	pos = origin - sphere->pos;
	abc = get_sphere_abc(sphere->radius, ray, pos);
	if (!solve_quadratic(abc.x, abc.y, abc.z, &res1, &res2))
		return (ret);
	else
	{
		if ((res1 < res2 && res1 > 0) || (res1 > res2 && res2 < 0))
			ret.dist = res1;
		else
			ret.dist = res2;
	}
	return (ret);
}


// OCL TO CUDA -> decoupe noe a test
__device__ t_ret	inter_sphere(t_sphere *sphere, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);
	float		res1 = 0.f;
	float		res2 = 0.f;
	float3		pos = make_float3(0.f);
	t_ret		ret;

	ret.dist = 0.f;
	ret.wall = 0;
	ret.normal = make_float3(0.f);
	pos = origin - sphere->pos;
	abc = get_sphere_abc(sphere->radius, ray, pos);
	if (!solve_quadratic(abc.x, abc.y, abc.z, &res1, &res2))
		return (ret);
	if (sphere->flags & OBJ_FLAG_PLANE_LIMIT)
		return (object_limited((t_object *)sphere, res1, res2, ray, origin));
	else
	{
		if ((res1 < res2 && res1 > 0) || (res1 > res2 && res2 < 0))
			ret.dist = res1;
		else
			ret.dist = res2;
	}
	if (sphere->flags & OBJ_FLAG_CUT)
		return(sphere_cut(sphere, ray, origin, res1, res2));
	return (ret);
}

// OCL TO CUDA -> OK
__device__ float3	get_cylinder_normal(t_cylinder *cylinder, t_hit hit)
{
	float3		res = make_float3(0.f);
	float3		v = make_float3(0.f);;
	float3		project = make_float3(0.f);;
	float		doty = 0.f;

	v = hit.pos - cylinder->pos;
	doty = dot(v, cylinder->dir);
	project = doty * cylinder->dir;
	res = v - project;
	return (normalize(res));
}

// OCL TO CUDA -> OK
__device__ float3	get_cylinder_abc(float radius, float3 dir, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);

	abc.x = dot(ray, ray) - (dot(ray, dir) * dot(ray, dir));
	abc.y = 2 * (dot(ray, origin) - (dot(ray, dir) * dot(origin, dir)));
	abc.z = dot(origin, origin) - (dot(origin, dir) * dot(origin, dir)) - (radius * radius);
	return (abc);
}

// OCL TO CUDA -> need test avec la decoupe de Noe
__device__ t_ret	inter_cylinder(t_cylinder *cylinder, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);
	float3		pos = make_float3(0.f);
	float		res1 = 0.f;
	float		res2 = 0.f;
	t_ret		ret;

	ret.dist = 0;
	ret.wall = 0;
	ret.normal = make_float3(0.f);
	pos = origin - cylinder->pos;
	abc = get_cylinder_abc(cylinder->radius, cylinder->dir, ray, pos);
	if (!solve_quadratic(abc.x, abc.y, abc.z, &res1, &res2))
		return (ret);
	if (cylinder->flags & OBJ_FLAG_PLANE_LIMIT)
		return (object_limited((t_object *)cylinder, res1, res2, ray, origin));
	else
	{
		if ((res1 < res2 && res1 > 0) || (res1 > res2 && res2 < 0))
			ret.dist = res1;
		else
			ret.dist = res2;
	}
	// if (cylinder->flags & OBJ_FLAG_CUT)
	// 	return(sphere_cut(cylinder, ray, origin, res1, res2));
	return (ret);
}

// OCL TO CUDA -> need test avec la decoupe de Noe
__device__ t_ret	inter_plan(t_plane *plane, float3 ray, float3 origin)
{
	float		t = 0.f;
	t_ret		ret;

	ret.dist = 0;
	ret.wall = 0;
	ret.normal = make_float3(0.f);
	t = dot(normalize(ray), plane->normal);
	if (fabsf(t) < EPSILONF || (plane->radius && t > plane->radius))
		return (ret);
	t = (dot(plane->pos - origin, plane->normal)) / t;
	if (t < EPSILONF)
		return (ret);
	if (plane->radius)
	{
		float3 p = origin + ray * t;
		float3 v = p - plane->pos;
		float d2 = dot(v, v);
		if (sqrtf(d2) > plane->radius)
			return (ret);
	}
	if (plane->flags & OBJ_FLAG_PLANE_LIMIT)
		return (object_limited((t_object *)plane, t, t, ray, origin));
	ret.dist = t;
	return (ret);
}

// OCL TO CUDA -> need test
__device__ float3		get_ellipsoid_normal(t_ellipsoid *ellipsoid, t_hit *hit)
{
	float3 pos = hit->pos - ellipsoid->pos;
	pos = vector_get_rotate(&pos, &ellipsoid->dir);

	float3 res = make_float3(0.f);

	res.x = (pos.x) / (ellipsoid->axis_size.x * ellipsoid->axis_size.x);
	res.y = (pos.y) / (ellipsoid->axis_size.y * ellipsoid->axis_size.y);
	res.z = (pos.z) / (ellipsoid->axis_size.z * ellipsoid->axis_size.z);
	res = vector_get_inverse(&res, &ellipsoid->dir);
	return (res);
}

// OCL TO CUDA -> need test avec la decoupe de Noe
__device__ t_ret	inter_ellipsoid(t_ellipsoid *ellipsoid, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);
	float		res1, res2 = 0.f;
	float3		pos = make_float3(0.f);
	float3		save_ray = make_float3(0.f);
	t_ret		ret;

	ret.dist = 0;
	ret.wall = 0;
	ret.normal = make_float3(0.f);
	save_ray = ray;
	pos = origin - ellipsoid->pos;
	pos = vector_get_rotate(&pos, &ellipsoid->dir);
	ray = vector_get_rotate(&ray, &ellipsoid->dir);
	ray = ray / ellipsoid->axis_size;
	pos = pos / ellipsoid->axis_size;

	abc.x = (ray.x * ray.x +
	 	ray.y * ray.y +
	 	ray.z * ray.z);
	abc.y = (2 * pos.x * ray.x +
		 2 * pos.y * ray.y +
		 2 * pos.z * ray.z);
	abc.z = (pos.x * pos.x +
		 pos.y * pos.y +
		 pos.z * pos.z) - (ellipsoid->radius * ellipsoid->radius);

	if (!solve_quadratic(abc.x, abc.y, abc.z, &res1, &res2))
		return (ret);
	if (ellipsoid->flags & OBJ_FLAG_PLANE_LIMIT)
		return (object_limited((t_object *)ellipsoid, res1, res2, save_ray, origin));
	if ((res1 < res2 && res1 > 0) || (res1 > res2 && res2 < 0))
		ret.dist = res1;
	else
		ret.dist = res2;
	return (ret);
}

// OCL TO CUDA -> need test
__device__ float3	get_cone_normal(t_cone *cone, t_hit hit)
{
	float3		res = make_float3(0.f);
	float3		v = make_float3(0.f);
	float3		project = make_float3(0.f);
	float		doty = 0.f;

	v = hit.pos - cone->pos;
	doty = dot(v, cone->dir);
	project = doty * cone->dir;
	res = v - project;

	return (normalize(res));
}

// OCL TO CUDA -> need test
__device__ float3	get_cone_abc(t_cone *cone, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);
	float		k = radians(cone->angle);

	k = tan(k);
	k = 1 + k * k;
	abc.x = dot(ray, ray) - (k * (dot(ray, cone->dir) * dot(ray, cone->dir)));
	abc.y = 2 * (dot(ray, origin) - (k * \
			(dot(ray, cone->dir) * dot(origin, cone->dir))));
	abc.z = (dot(origin, origin) - \
			(k * (dot(origin, cone->dir) * dot(origin, cone->dir))));
	return (abc);
}

// OCL TO CUDA -> need test avec decoupe Noe
__device__ t_ret	inter_cone(t_cone *cone, float3 ray, float3 origin)
{
	float3		abc = make_float3(0.f);
	float		res1 = 0.f;
	float		res2 = 0.f;
	float3		pos = make_float3(0.f);
	t_ret		ret;

	ret.dist = 0;
	ret.wall = 0;
	ret.normal = make_float3(0.f);
	pos = origin - cone->pos;
	abc = get_cone_abc(cone, ray, pos);
	if (!solve_quadratic(abc.x, abc.y, abc.z, &res1, &res2))
		return (ret);
	if ((res1 < res2 && res1 > 0) || (res1 > res2 && res2 < 0))
		ret.dist = res1;
	else
		ret.dist = res2;
	return (ret);
}

// OCL TO CUDA -> need test
__device__ unsigned int		cone_texture(float3 pos, float3 dir, float3 u_axis, unsigned int *texture, int t_width, int t_height, float3 ratio, float3 offset)
{
	unsigned int	color = 0;
	float3			v_axis = make_float3(0.f);
	float			npos = 0.f;
	float			vpos = 0.f;
	int2			uv = make_int2(0);

	v_axis = cross(u_axis, dir);
	npos = dot(pos, dir);
	while (npos > 10)
		npos -= 10;
	while (npos < 0)
		npos += 10;
	uv.y = (int)(floor((length(npos * dir) / 10) * ratio.y * (t_height - 1) + offset.y));
	npos = dot(pos, u_axis);
	vpos = dot(pos, v_axis);
	uv.x = (int)(floor((0.5 + (atan2(npos, vpos) / (2 * M_PI))) * ratio.x * (t_width - 1) + offset.x));
	uv.x %= t_width - 1;
	uv.y %= t_height - 1;
	if (uv.x < 0)
		uv.x = uv.x + t_width - 1;
	if (uv.y < 0)
		uv.y = -uv.y;
	else
		uv.y = (uv.y - t_height - 1) * -1;
	uv.x %= t_width - 1;
	uv.y %= t_height - 1;
	 color = (unsigned int)texture[uv.x + (uv.y * t_width)];
	return (color);
}


// OCL TO CUDA -> need test
__device__ t_hit		ray_hit(t_scene *scene, float3 origin, float3 ray, float lightdist)
{
	t_hit						hit;
	t_object 					*obj;
	unsigned int				mem_index_obj = 0;
	t_ret						ret;

	hit = hit_init();
	obj = 0;
	ret.dist = 0.f;
	ret.wall = 0;
	ret.normal = make_float3(0.f);
	if (lightdist == 0)
		hit.opacity = 1;
	while (mem_index_obj < scene->mem_size_obj)
	{
		obj = (t_object *)((char *)scene->mem_obj + mem_index_obj);
		if (obj->type == OBJ_SPHERE)
		 	ret = inter_sphere((struct s_sphere *)obj, ray, origin);
		else if (obj->type == OBJ_CYLINDER)
		 	ret = inter_cylinder((struct s_cylinder *)obj, ray, origin);
		else if (obj->type == OBJ_PLANE)
		 	ret = inter_plan((struct s_plane *)obj, ray, origin);
		else if (obj->type == OBJ_CONE)
		 	ret = inter_cone((struct s_cone *)obj, ray, origin);
		else if (obj->type == OBJ_ELLIPSOID)
		   	ret = inter_ellipsoid((struct s_ellipsoid *)obj, ray, origin);
		else if (obj->type == OBJ_THOR)
			ret = inter_thor((struct s_thor *)obj, ray, origin);
		else if (obj->type == OBJ_KUBE)
			ret = inter_kube((struct s_kube *)obj, ray, origin);
		if (lightdist > 0 && ret.dist < lightdist && ret.dist > EPSILON)
			hit.opacity += obj->opacity;
		if ((ret.dist < hit.dist || hit.dist == 0) && ret.dist > EPSILON)
		{
			hit.dist = ret.dist;
			hit.normal = ret.normal;
			hit.wall = ret.wall;
			hit.obj = obj;
			hit.lock = 1;
			hit.mem_index = mem_index_obj;
		}
		mem_index_obj += obj->size;
	}
	return (hit);
}

// OCL TO CUDA -> need test
__device__ float3		get_hit_normal(t_scene *scene, float3 ray, t_hit hit)
{
	float3						res, save = make_float3(0.f);
	t_object 					*object;

	object = hit.obj;
	res = make_float3(0.f);
	if (hit.wall)
		res = hit.normal;
	else
	{
		if (hit.obj->type == OBJ_SPHERE)
		{ 
			if (hit.obj->flags & OBJ_FLAG_CUT)
			{
				t_ret tmp  = mini_inter_sphere((t_sphere*)hit.obj, ray, ACTIVECAM.pos);
				if (tmp.dist < hit.dist)
					res = hit.obj->pos - hit.pos;
				else
					res = hit.pos - hit.obj->pos;
			}
			else
				res = hit.pos - hit.obj->pos;
		}
		else if (hit.obj->type == OBJ_CYLINDER)
			res = get_cylinder_normal((t_cylinder *)hit.obj, hit);
		else if (hit.obj->type == OBJ_CONE)
			res = get_cone_normal((t_cone *)hit.obj, hit);
		else if (hit.obj->type == OBJ_ELLIPSOID)
			res = get_ellipsoid_normal(( t_ellipsoid *)hit.obj, &hit);
		else if (hit.obj->type == OBJ_THOR)
			res = get_thor_normal((t_thor *)hit.obj, hit.pos);
		else if (hit.obj->type == OBJ_KUBE)
			res = get_kube_normal((t_kube *)hit.obj, hit.pos);
		else if (hit.obj->type == OBJ_PLANE)
		{
			if (dot(hit.obj->dir, ray * -1) < 0)
				res = hit.obj->dir * -1;
			else
				res = hit.obj->dir;
		}
	}
	save = res;
	if (object->flags & OBJ_FLAG_WAVES)
	{
		if (object->type == OBJ_PLANE)
			save.y = res.y + object->waves_p1.x * sinf((hit.pos.x + scene->u_time));
		// no sinwave with torus
		else if (object->type == OBJ_THOR)
		{
			save.x = res.x + object->waves_p1.x * 100 * sinf(res.y * object->waves_p2.x + scene->u_time);
			save.z = res.z + object->waves_p1.y * 100 * sinf(res.x * object->waves_p2.y + scene->u_time);
			save.y = res.y + object->waves_p1.z * 100 * sinf(res.x * object->waves_p2.z + scene->u_time);
		}
		else
		{
			save.x = res.x + object->waves_p1.x * sinf(res.y * object->waves_p2.x + scene->u_time);
			save.z = res.z + object->waves_p1.y * sinf(res.x * object->waves_p2.y + scene->u_time);
			save.y = res.y + object->waves_p1.z * sinf(res.x * object->waves_p2.z + scene->u_time);
		}
	}

	return (normalize(save));
}

// OCL TO CUDA -> need test
__device__ unsigned int		cylinder_texture(float3 pos, t_cylinder *cyl, unsigned int *texture, int t_width, int t_height)
{
	unsigned int	color = 0;
	float3			v_axis = make_float3(0.f);
	float			npos = 0;
	float			vpos = 0;
	int2			uv = make_int2(0);

	v_axis = cross(cyl->u_axis, cyl->dir);
	npos = length(dot(pos, cyl->dir) * cyl->dir);
	while (npos > 10)
		npos -= 10;
	if (dot(pos, cyl->dir) < 0)
		npos = (npos - 10) * -1;
	uv.y = (int)(floor((npos / 10) * t_height * cyl->diff_ratio.y + cyl->diff_offset.y));
	npos = dot(pos, cyl->u_axis);
	vpos = dot(pos, v_axis);
	uv.x = (int)(floor((0.5 + (atan2(npos, vpos) / (2 * M_PI))) * t_width * cyl->diff_ratio.x + cyl->diff_offset.x));
	uv.x %= t_width;
	uv.y %= t_height;
	if (uv.x < 0)
		uv.x = uv.x + t_width;
	if (uv.y < 0)
		uv.y = -uv.y;
	else
		uv.y = (uv.y - t_height) * -1;
	uv.x %= t_width;
	uv.y %= t_height;
	color = (unsigned int)texture[uv.x + (uv.y * t_width)];
	return (color);
}

// OCL TO CUDA -> need test
__device__ unsigned int			phong(t_scene *scene, t_hit hit, float3 ray)
{
	t_object 			*obj;
	t_light 			*light;
	unsigned int		mem_index_lights = 0;

	unsigned int		res_color = 0;
	float				tmp = 0.f;
	float3				reflect = make_float3(0.f);
	float3 				diffuse = make_float3(0.f);
	float 				brightness = 0.f;
	unsigned int 		hue = 0;
	unsigned int 		hue_light = 0;
	unsigned int 		col_r, col_g, col_b, obj_r, obj_g, obj_b, l_r, l_b, l_g = 0;
	t_light_ray			light_ray;
	t_hit				light_hit;
	float 				pow_of_spec = 0.f;
	int 				light_color = 0;
	float3 			speculos = make_float3(0.f);

	obj = hit.obj;
	if ((hit.obj->flags & OBJ_FLAG_CHECKERED) || hit.obj->flags & OBJ_FLAG_DIFF_MAP)
		hue = hit.color;
	else
		hue = obj->color;

	col_r = (hue & 0x00FF0000) >> 16;
	col_g = (hue & 0x0000FF00) >> 8;
	col_b = (hue & 0x000000FF);
	col_r = (0.01 + col_r * scene->ambient.x > 255 ? 255 : 0.01 + col_r * scene->ambient.x);
	col_g = (0.01 + col_g * scene->ambient.y > 255 ? 255 : 0.01 + col_g * scene->ambient.y);
	col_b = (0.01 + col_b * scene->ambient.z > 255 ? 255 : 0.01 + col_b * scene->ambient.z);
	res_color = ((col_r << 16) + (col_g << 8) + col_b);

	while (mem_index_lights < scene->mem_size_lights)
	{
		tmp = 0;
		light = (t_light *)((char *)scene->mem_lights + mem_index_lights);
		light_ray.dir = light->pos - hit.pos;
		light_ray.dist = length(light_ray.dir);
		light_ray.dir = normalize(light_ray.dir);
		light_hit = ray_hit(scene, hit.pos, light_ray.dir, light_ray.dist);
		if (!(light_hit.dist < light_ray.dist && light_hit.dist > EPSILON) || (light_hit.opacity < 1 && scene->depth != 0))
		{
			// diffuse part
			tmp = (dot(hit.normal, light_ray.dir));
			if (tmp > EPSILON)
			{
				brightness = (float )light->brightness;
				diffuse = (float3 )obj->diff;
				hue_light = light->color;

				col_r = (res_color & 0xFF0000) >> 16;
				col_g = (res_color & 0x00FF00) >> 8;
				col_b = (res_color & 0x0000FF);
				obj_r = (hue & 0xFF0000) >> 16;
				obj_g = (hue & 0x00FF00) >> 8;
				obj_b = (hue & 0x00000FF);
				l_r = (hue_light & 0xFF0000) >> 16;
				l_g = (hue_light & 0x00FF00) >> 8;
				l_b = (hue_light & 0x0000FF);

				// col_r += ((l_r * brightness) + obj_r) * tmp * diffuse.x;
				// col_g += ((l_g * brightness) + obj_g) * tmp * diffuse.y;
				// col_b += ((l_b * brightness) + obj_b) * tmp * diffuse.z;
				col_r += (((l_r * brightness) * obj_r) * tmp * diffuse.x) / 255.0;
				col_g += (((l_g * brightness) * obj_g) * tmp * diffuse.y) / 255.0;
				col_b += (((l_b * brightness) * obj_b) * tmp * diffuse.z) / 255.0;

				(col_r > 255 ? col_r = 255 : 0);
				(col_g > 255 ? col_g = 255 : 0);
				(col_b > 255 ? col_b = 255 : 0);
				
				res_color = ((col_r << 16) + (col_g << 8) + col_b);
			}

			// specular part
			reflect = normalize(((float)(2.0f * dot(hit.normal, light_ray.dir)) * hit.normal) - light_ray.dir);
			tmp = dot(reflect, ray * -1);
			if (tmp > EPSILON)
			{
				speculos = obj->spec;
				col_r = (res_color & 0x00FF0000) >> 16;
				col_g = (res_color & 0x0000FF00) >> 8;
				col_b = (res_color & 0x000000FF);

				pow_of_spec = powf(tmp, (light->shrink));
				light_color = light->color;

				col_r += (((light_color & 0xFF0000) >> 16) * pow_of_spec) * speculos.x;
				col_g += ((light_color & 0x00FF00) >> 8) * pow_of_spec * speculos.y;
				col_b += (light_color & 0x0000FF) * pow_of_spec * speculos.z;

				(col_r > 255 ? col_r = 255 : 0);
				(col_g > 255 ? col_g = 255 : 0);
				(col_b > 255 ? col_b = 255 : 0);

				res_color = ((col_r << 16) + (col_g << 8) + col_b);
			}


			// opacite de l'ombre à debug!!!!!
			res_color = blend_factor(res_color, ((light_hit.opacity - 1) * -1));
			// res_color = blend_factor(res_color, 1 - (light_hit.opacity / 2));
	
			// si pas de depth alors opacity = 0, la couleur n'est pas touchée
			// res_color = blend_factor(res_color, 1 - (light_hit.opacity * light_hit.opacity));
			// if (light_hit.opacity != 0)
			// {
			// 	light_hit.opacity = 1 - light_hit.opacity;
			// 	if (light_hit.opacity > scene->ambient.x)
			// 		col_r = (0.01 + col_r * light_hit.opacity > 255 ? 255 : 0.01 + col_r * light_hit.opacity);
			// 	else
			// 		col_r = (0.01 + col_r * scene->ambient.x > 255 ? 255 : 0.01 + col_r * scene->ambient.x);
			// 	if (light_hit.opacity > scene->ambient.y)
			// 		col_g = (0.01 + col_g * light_hit.opacity > 255 ? 255 : 0.01 + col_g * light_hit.opacity);
			// 	else
			// 		col_g = (0.01 + col_g * scene->ambient.y > 255 ? 255 : 0.01 + col_g * scene->ambient.y);
			// 	if (light_hit.opacity > scene->ambient.z)
			// 		col_b = (0.01 + col_b * light_hit.opacity > 255 ? 255 : 0.01 + col_b * light_hit.opacity);					
			// 	else
			// 		col_b = (0.01 + col_b * scene->ambient.z > 255 ? 255 : 0.01 + col_b * scene->ambient.z);
			// 	res_color = ((col_r << 16) + (col_g << 8) + col_b);
			// }


			if (scene->flag & OPTION_CARTOON_FOUR)
			 	res_color = cartoonize_four(col_r, col_g, col_b);
			else if (scene->flag & OPTION_CARTOON_TWO)
			 	res_color = cartoonize_two(col_r, col_g, col_b);
		}
		mem_index_lights += light->size;
	}
	return (res_color);
}

// OCL TO CUDA -> need test
__device__ float		reflect_ratio(float n1, float n2, float cos1, float sint)
{
	float			fr1 = 0.f;
	float			fr2 = 0.f;
	float			cos2 = sqrtf(1 - sint * sint);

	if (cos1 >= 0)
	{
		fr1 = n1;
		n1 = n2;
		n2 = fr1;
	}
	else
		cos1 = -cos1;
	if (n1 / n2 * sqrtf(1 - cos1 * cos1) > 1)
		return (1);
	fr1 = (n2 * cos1 - n1 * cos2) / (n2 * cos1 + n1 * cos2);
	fr2 = (n1 * cos2 - n2 * cos1) / (n1 * cos2 + n2 * cos1);
	fr1 *= fr1;
	fr2 *= fr2;
	return ((fr1 + fr2) / 2);
}

// OCL TO CUDA -> need test
__device__ float3		refract_ray(t_scene *scene, float3 ray, float3 normale, float tra)
{
	float3			refract = make_float3(0.f);
	float			c1 = 0.f;
	float			c2 = 0.f;
	float			eta = 0.f;

	c1 = dot(normale, ray);
	eta = 1 / tra;
	if (c1 < 0)
		c1 = -c1;
	else
	{
		normale = -normale;
		eta = tra;
	}
	c2 = sqrtf(1 - ((eta * eta) * (1 - (c1 * c1))));
	// DEUXIEME LOIS DE SNELL-DECARTES /////////////////////////////////////////////
	refract = normalize((eta * ray) + ((eta * c1) - c2) * normale);
	////////////////////////////////////////////////////////////////////////////////
	return (refract);
}

__device__ float3		bounce_ray(t_scene *scene, float3 ray, t_tor tor)
{
	float3			reflex;

	reflex = make_float3(0.f);
	// PREMIÈRE LOI DE SNELL-DESCARTES ///////////////////////////////////////////////////////////
	reflex = normalize(ray - (2 * (float)dot(tor.normale, ray) * tor.normale));
	//////////////////////////////////////////////////////////////////////////////////////////////
	return (reflex);
}

// OCL TO CUDA -> need test
__device__ int			tor_height(int i)
{
	int				h = 0;

	while ((i = (i - 1) / 2) >= 0)
		h++;
	return (h);
}

// OCL TO CUDA -> need test
__device__ unsigned int	tor_final_color(t_tor *tor)
{
	int				i = 31;
	unsigned int	color = 0;

	while (i > 0)
	{
		if (tor[i].activate == 0 || (tor[(2 * i) + 2].activate == 0 && tor[(2 * i) + 1].activate == 0))
			;
		else
		{
			color = blend_add(blend_factor(tor[(2 * i) + 2].color, tor[(2 * i) + 2].ratio), blend_factor(tor[(2 * i) + 1].color, tor[(2 * i) + 1].ratio));
			if (tor[i].coef_tra != 0)
				tor[i].color = blend_add(blend_factor(tor[i].color, tor[i].opacity), blend_factor(color, 1 - tor[i].opacity));
			else if (tor[i].coef_ref != 0)
				tor[i].color = blend_add(blend_factor(tor[i].color, 1 - tor[i].coef_ref), blend_factor(color, tor[i].coef_ref));
		}
		i = i - 1;
	}
	color = blend_add(blend_factor(tor[(2 * i) + 2].color, tor[(2 * i) + 2].ratio), blend_factor(tor[(2 * i) + 1].color, tor[(2 * i) + 1].ratio));
	if (tor[i].coef_tra != 0)
		color = blend_add(blend_factor(tor[i].color, tor[i].opacity), blend_factor(color, 1 - tor[i].opacity));
	else
		color = blend_add(blend_factor(tor[i].color, 1 - tor[i].coef_ref), blend_factor(color, tor[i].coef_ref));
	return (color);
}

// OCL TO CUDA -> need test
__device__ t_tor		tor_push(float3 ray, float3 normale, float3 pos, float coef_ref, float coef_tra, float opacity, unsigned int color, int type, float ratio)
{
	t_tor			tor;

	tor.prim = ray;
	tor.pos = pos;
	tor.normale = normale;
	tor.coef_ref = coef_ref;
	tor.coef_tra = coef_tra;
	tor.opacity = opacity;
	tor.color = color;
	tor.activate = 1;
	tor.ratio = ratio;
	tor.type = type;
	return (tor);
}

// OCL TO CUDA -> need test
__device__ unsigned int	fresnel(t_scene *scene, float3 ray, t_hit old_hit, int depth, unsigned int color)
{
	t_hit			new_hit;
	unsigned int	ncolor = 0;
	float3			new_ray = make_float3(0.f);
	float3			dummy = make_float3(0.f);
	float			fr = 0.f;
	float			cos1 = 0.f;
	float			sint = 0.f;
	t_tor			tor[63];
	int				i = 0;
	int				tor_depth = 0;

	tor_depth = (int)(pow(2.f, (float)(depth))) - 1;
	i = 0;
	while (i < 63)
	{
		tor[i].activate = 0;
		tor[i].prim = make_float3(0.f);
		tor[i].normale = make_float3(0.f);
		tor[i].pos = make_float3(0.f);
		tor[i].coef_ref = 0.f;
		tor[i].coef_tra = 0.f;
		tor[i].color = 0;
		tor[i].opacity = 0.f;
		tor[i].type = 0;
		tor[i].ratio = 0.f;
		i++;
	}
	i = 0;
	
	tor[i] = tor_push(	ray, 
						old_hit.normal, 
						old_hit.pos, 
						old_hit.obj->reflex,
						old_hit.obj->refract, 
						old_hit.obj->opacity, 
						color,
						old_hit.obj->type, 
						0);

	while (i < 31 && i < tor_depth)
	{
		int rhododendron = 0;
		if (tor[i].coef_tra != 0)
		{
			while (rhododendron < 4)
			{
				cos1 = dot(tor[i].normale, tor[i].prim);
				if (cos1 >= 0)
					sint = tor[i].coef_tra * sqrt(1 - cos1 * cos1);
				else
					sint = 1 / tor[i].coef_tra * sqrt(1 - cos1 * cos1);
				if (sint >= 1)
					fr = 1;
				else
					fr = reflect_ratio(1, tor[i].coef_tra, cos1, sint);
				if (fr < 1)
				{
					if (tor[i].type != OBJ_PLANE)
						new_ray = refract_ray(scene, tor[i].prim, tor[i].normale, tor[i].coef_tra);
					else
						new_ray = tor[i].prim;
					if (cos1 < 0)
						new_hit = ray_hit(scene, tor[i].pos + (0.001f * (2.f * -tor[i].normale)), new_ray, 0);
					else
						new_hit = ray_hit(scene, tor[i].pos, new_ray, 0);
					if (new_hit.dist > 0 && new_hit.dist < MAX_DIST)
					{
						if (cos1 < 0)
							new_hit.pos = (new_hit.dist * new_ray) + tor[i].pos + (0.001f * (2.f * -tor[i].normale));
						else
							new_hit.pos = (new_hit.dist * new_ray) + tor[i].pos;
						new_hit.normal = get_hit_normal(scene, new_ray, new_hit);
						new_hit.pos = new_hit.pos + (0.001f * new_hit.normal);

						if ((new_hit.obj->type == OBJ_SPHERE) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
							new_hit.color = sphere_texture(normalize(new_hit.obj->pos - new_hit.pos), scene->texture_earth, 4915, 2457, ((t_sphere *)new_hit.obj)->diff_ratio, ((t_sphere *)new_hit.obj)->diff_offset);
						else if ((new_hit.obj->type == OBJ_SPHERE) && (new_hit.obj->flags & OBJ_FLAG_CHECKERED))
							new_hit.color = sphere_checkerboard(normalize(new_hit.obj->pos - new_hit.pos), new_hit.obj->color, new_hit.obj->check_size);

						else if ((new_hit.obj->type == OBJ_PLANE) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
							new_hit.color = plane_texture(new_hit.normal, new_hit.pos, ((t_plane *)new_hit.obj)->u_axis, ((t_plane *)new_hit.obj)->diff_ratio, ((t_plane *)new_hit.obj)->diff_offset, scene->texture_star, 1500, 1500);
						else if ((new_hit.obj->type == OBJ_PLANE) && (new_hit.obj->flags & OBJ_FLAG_CHECKERED))
							new_hit.color = plane_checkerboard(new_hit.normal, new_hit.pos, new_hit.obj->color, new_hit.obj->check_size);

						else if ((new_hit.obj->type == OBJ_CYLINDER) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
							new_hit.color = cylinder_texture(new_hit.pos - new_hit.obj->pos, (t_cylinder *)new_hit.obj, scene->texture_star, 1500, 1500);

						else if ((new_hit.obj->type == OBJ_CONE) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
							new_hit.color = cone_texture(new_hit.pos - new_hit.obj->pos, new_hit.obj->dir, ((t_cone *)new_hit.obj)->u_axis, scene->texture_star, 1500, 1500, ((t_cone *)new_hit.obj)->diff_ratio, ((t_cone *)new_hit.obj)->diff_offset);
						else
							new_hit.color = new_hit.obj->color;
		
						ncolor = phong(scene, new_hit, new_ray);
						tor[(i * 2) + 1] = tor_push(new_ray, new_hit.normal, new_hit.pos, new_hit.obj->reflex, new_hit.obj->refract, new_hit.obj->opacity, ncolor, new_hit.obj->type, 1 - fr);
					}
					else
					{
						if (scene->flag & OPTION_SKYBOX)
							ncolor = skybox(new_ray, scene->texture_star, 4096, 2048);
						else
							ncolor = get_ambient(scene, BACKCOLOR);
						tor[(i * 2) + 1] = tor_push(new_ray, new_hit.normal, new_hit.pos, 0, 0, 0, ncolor, 0, 1 - fr);
					}
				}
				rhododendron++;
				i++;
			}
			i -= 4;
		}
		else if (tor[i].coef_ref != 0)
			fr = 1;
		else
			fr = 0;
		if (fr > 0)
		{
			cos1 = dot(tor[i].normale, tor[i].prim);
			if (cos1 >= 0)
				new_ray = bounce_ray(scene, tor[i].prim, tor[i]);
			else
				new_ray = bounce_ray(scene, tor[i].prim, tor[i]);
			if (cos1 >= 0)
				new_hit = ray_hit(scene, tor[i].pos + (0.001f * (2.f * -tor[i].normale)), new_ray, 0);
			else
				new_hit = ray_hit(scene, tor[i].pos, new_ray, 0);
			if (new_hit.dist > 0 && new_hit.dist < MAX_DIST)
			{
				if (cos1 >= 0)
					new_hit.pos = (new_hit.dist * new_ray) + tor[i].pos + (0.001f * (2.f * -tor[i].normale));
				else
					new_hit.pos = (new_hit.dist * new_ray) + tor[i].pos;
				new_hit.normal = get_hit_normal(scene, new_ray, new_hit);
				new_hit.pos = new_hit.pos + (new_hit.dist / SHADOW_BIAS * new_hit.normal);


				if ((new_hit.obj->type == OBJ_SPHERE) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
					new_hit.color = sphere_texture(normalize(new_hit.obj->pos - new_hit.pos), scene->texture_earth, 4915, 2457, ((t_sphere *)new_hit.obj)->diff_ratio, ((t_sphere *)new_hit.obj)->diff_offset);
				else if ((new_hit.obj->type == OBJ_SPHERE) && (new_hit.obj->flags & OBJ_FLAG_CHECKERED))
					new_hit.color = sphere_checkerboard(normalize(new_hit.obj->pos - new_hit.pos), new_hit.obj->color, new_hit.obj->check_size);

				else if ((new_hit.obj->type == OBJ_PLANE) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
					new_hit.color = plane_texture(new_hit.normal, new_hit.pos, ((t_plane *)new_hit.obj)->u_axis, ((t_plane *)new_hit.obj)->diff_ratio, ((t_plane *)new_hit.obj)->diff_offset, scene->texture_star, 1500, 1500);
				else if ((new_hit.obj->type == OBJ_PLANE) && (new_hit.obj->flags & OBJ_FLAG_CHECKERED))
					new_hit.color = plane_checkerboard(new_hit.normal, new_hit.pos, new_hit.obj->color, new_hit.obj->check_size);

				else if ((new_hit.obj->type == OBJ_CYLINDER) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
					new_hit.color = cylinder_texture(new_hit.pos - new_hit.obj->pos, (t_cylinder *)new_hit.obj, scene->texture_star, 1500, 1500);

				else if ((new_hit.obj->type == OBJ_CONE) && (new_hit.obj->flags & OBJ_FLAG_DIFF_MAP))
					new_hit.color = cone_texture(new_hit.pos - new_hit.obj->pos, new_hit.obj->dir, ((t_cone *)new_hit.obj)->u_axis, scene->texture_star, 1500, 1500, ((t_cone *)new_hit.obj)->diff_ratio, ((t_cone *)new_hit.obj)->diff_offset);
				else
					new_hit.color = new_hit.obj->color;
				
				ncolor = phong(scene, new_hit, new_ray);
				tor[(2 * i) + 2] = tor_push(new_ray, new_hit.normal, new_hit.pos, new_hit.obj->reflex, new_hit.obj->refract, new_hit.obj->opacity, ncolor, new_hit.obj->type, fr);
			}
			else
			{
				if (scene->flag & OPTION_SKYBOX)
					ncolor = skybox(new_ray, scene->texture_star, 4096, 2048);
				else
					ncolor = get_ambient(scene, BACKCOLOR);
				tor[(2 * i) + 2] = tor_push(new_ray, new_hit.normal, new_hit.pos, 0, 0, 0, ncolor, 0, fr);

			}
		}
		i = i + 1;
		while (i < 31 && tor[i].activate == 0)
			i = i + 1;
	}
	return (tor_final_color(tor));
}

// OCL TO CUDA -> need test
__device__ unsigned int	get_pixel_color(t_scene *scene, float3 ray, int *target, bool isHim, int index)
{
	t_hit			hit;
	int				depth = 0;
	unsigned int	color = 0;
	unsigned int	bounce_color = 0;

	hit = hit_init();
	hit.dist = MAX_DIST;
	depth = scene->depth;
	
	// DEBUG
	if (depth < 0)
		printf("!WARNING!\nget_pixel_color | depth < 0 !!!!\n");

	hit = ray_hit(scene, (ACTIVECAM.pos), ray, 0);
	if ((isHim == 1) && (hit.lock == 1))
		*target = hit.mem_index;
	if (hit.dist > EPSILON && hit.dist < MAX_DIST)
	{
		hit.pos = (hit.dist * ray) + (ACTIVECAM.pos);
		hit.normal = get_hit_normal(scene, ray, hit);
		hit.pos = hit.pos + (hit.dist / 1000.f * hit.normal);


		if ((hit.obj->type == OBJ_SPHERE) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			 hit.color = sphere_texture(normalize(hit.obj->pos - hit.pos), scene->texture_earth, 4915, 2457, ((t_sphere *)hit.obj)->diff_ratio, ((t_sphere *)hit.obj)->diff_offset);
		else if ((hit.obj->type == OBJ_SPHERE) && (hit.obj->flags & OBJ_FLAG_CHECKERED))
			hit.color = sphere_checkerboard(normalize(hit.obj->pos - hit.pos), hit.obj->color, hit.obj->check_size);

		else if ((hit.obj->type == OBJ_PLANE) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			hit.color = plane_texture(hit.normal, hit.pos, ((t_plane *)hit.obj)->u_axis, ((t_plane *)hit.obj)->diff_ratio, ((t_plane *)hit.obj)->diff_offset, scene->texture_star, 1500, 1500);
		else if ((hit.obj->type == OBJ_PLANE) && (hit.obj->flags & OBJ_FLAG_CHECKERED))
			hit.color = plane_checkerboard(hit.normal, hit.pos, hit.obj->color, hit.obj->check_size);

		else if ((hit.obj->type == OBJ_CYLINDER) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			hit.color = cylinder_texture(hit.pos - hit.obj->pos, (t_cylinder *)hit.obj, scene->texture_star, 1500, 1500);

		else if ((hit.obj->type == OBJ_CONE) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			hit.color = cone_texture(hit.pos - hit.obj->pos, hit.obj->dir, ((t_cone *)hit.obj)->u_axis, scene->texture_star, 1500, 1500, ((t_cone *)hit.obj)->diff_ratio, ((t_cone *)hit.obj)->diff_offset);

		color = phong(scene, hit, ray);
		if (((hit.obj->refract != 0 && hit.obj->opacity < 1) || hit.obj->reflex > 0) && depth > 0)
			return (fresnel(scene, ray, hit, depth, color));

		return (blend_add(color, bounce_color));
	}
	
	if (scene->flag & OPTION_SKYBOX)
	{
		color = skybox(ray, scene->texture_star, 4096, 2048);
		return (color);
	}
	else
		return (get_ambient(scene, BACKCOLOR));
}


// OCL TO CUDA -> ok
__device__ float3		get_ray_cam(t_scene *scene, int x, int y, int width, int height)
{
	float3			cam_ray = make_float3(0.f);
	float			ratio = ((float)width) / ((float)height);

	cam_ray.x = ((2.f * (((float)x + 0.5f) / (float)width)) - 1.f) * ratio * (tanf(radians(ACTIVECAM.fov / 2.f)));
	cam_ray.y = ((1.f - (2 * (((float)y + 0.5f) / (float)height))) * tanf(radians(ACTIVECAM.fov / 2.f)));
	cam_ray.z = 1.f;
	cam_ray = rotat_zyx(cam_ray, ACTIVECAM.pitch, ACTIVECAM.yaw, 0);
	return(normalize(cam_ray));
}

__device__ unsigned int	ray_trace(	int				index,
									char			*mem_objects,
									int				mem_size_objects,
									float			u_time,

									t_scene			*scene,
									t_cam			*cameras,

									char			*mem_lights,
									int				mem_size_lights,

									int				*target,
									unsigned int	*texture_earth,
									unsigned int	*texture_moon,
									unsigned int	*texture_earth_cloud,
									unsigned int	*texture_star)
{
	unsigned int	final_color = 0;
	unsigned int	final_color_o[32] = {0};
	uint3			rgb = make_uint3(0);
	float3			prim_ray = make_float3(0.f);
	int				x = 0;
	int				y = 0;

	x = index % scene->win_w;
	y = index / scene->win_w;

	scene->cameras = cameras;
	scene->mem_lights = mem_lights;
	scene->u_time = u_time;
	scene->mem_obj = mem_objects;
	scene->mem_size_obj = mem_size_objects;
	scene->mem_size_lights = mem_size_lights;
	scene->texture_earth = texture_earth;
	scene->texture_earth_cloud = texture_earth_cloud;
	scene->texture_moon = texture_moon;
	scene->texture_star = texture_star;
	
	if (scene->flag & OPTION_RUN && scene->mou_x + scene->win_h * scene->mou_y == index)
		*target = -1;
	
	final_color = 0;

	if (scene->over_sampling > 1)
	{
		uint2 true_pix = make_uint2(x, y);
		unsigned int lap = 0;
		x *= scene->over_sampling;
		y *= scene->over_sampling;

		while (lap < scene->over_sampling * 2)
		{
			x += (lap % 2);
			y += !(lap % 2);
			prim_ray = get_ray_cam(scene, x, y, scene->win_w * scene->over_sampling, scene->win_h * scene->over_sampling);
			final_color_o[lap] = get_pixel_color(scene, prim_ray, target, (scene->flag & OPTION_RUN && true_pix.x == scene->mou_x && true_pix.y == scene->mou_y), index);
			rgb.x += (final_color_o[lap] & 0x00FF0000);
			rgb.y += (final_color_o[lap] & 0x0000FF00);
			rgb.z += (final_color_o[lap] & 0x000000FF);
			lap++;
		}
		final_color += ((rgb.x / (scene->over_sampling * 2)) & 0x00FF0000);
		final_color += ((rgb.y / (scene->over_sampling * 2)) & 0x0000FF00);
		final_color += ((rgb.z / (scene->over_sampling * 2)) & 0x000000FF);
	}
	else
	{
		prim_ray = get_ray_cam(scene, x, y, scene->win_w, scene->win_h);
		final_color = get_pixel_color(scene, prim_ray, target, (scene->flag & OPTION_RUN && x == scene->mou_x && y == scene->mou_y), index);
	}

	if (scene->flag & OPTION_SEPIA)
		final_color = sepiarize(final_color);
	if (scene->flag & OPTION_BW)
		final_color = desaturate(final_color);
	if (scene->flag & OPTION_INVERT)
		final_color = invert(final_color);

	// test
	__syncthreads();

	// ALPHA INSERT and RGB SWAP
	int nw, nx, ny, nz;
	nw = 255;
	nx = (final_color & 0x00FF0000) >> 16;
	ny = (final_color & 0x0000FF00) >> 8;
	nz = (final_color & 0x000000FF);
	final_color = ((nw << 24) + (nz << 16) + (ny << 8) + nx);
	return (final_color);
}

__global__ void rt_launcher(unsigned int *output,
						char *mem_objects, int mem_size_objects,
						float u_time,
						t_scene *scene_data, t_cam *cameras_data,
						char *mem_lights, int mem_size_lights, int *target,
						unsigned int *texture_0,
						unsigned int *texture_1,
						unsigned int *texture_2,
						unsigned int *texture_3)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * scene_data->win_w + col;
	output[index] = ray_trace(index, mem_objects, mem_size_objects,
								u_time,
							scene_data, cameras_data,
							mem_lights, mem_size_lights, target, texture_0, texture_1, texture_2, texture_3);
}

extern "C" void render_cuda(t_cuda			*cuda,
							int 			*pixel_data,
							int				*target,
							t_gen			*gen_objects,
							t_gen			*gen_lights,
							float			u_time,
							t_scene			*scene_data,
							t_cam			*cameras_data)
{
	dim3					threads_per_block(8, 8);
	dim3					grid_size(scene_data->win_w / threads_per_block.x, scene_data->win_h / threads_per_block.y);
		/*printf("GPU\n");
		printf("t_cam %zu\n", sizeof(t_cam));
		printf("t_scene %zu\n", sizeof(t_scene));
		printf("t_object %zu\n", sizeof(t_object));
		printf("t_gen %zu\n", sizeof(t_gen));
		printf("t_sphere %zu\n", sizeof(t_sphere));
		printf("t_light %zu\n", sizeof(t_light));
		printf("t_cylinder %zu\n", sizeof(t_cylinder));
		printf("t_sphere %zu\n", sizeof(t_sphere));
		printf("t_ellipsoid %zu\n", sizeof(t_ellipsoid));
		printf("t_plane %zu\n", sizeof(t_plane));
		printf("t_cone %zu\n", sizeof(t_cone));
		printf("t_cone %zu\n", sizeof(t_cone));
		printf("\n");*/

	hipMemcpy(cuda->mem[1], gen_objects->mem, gen_objects->mem_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda->mem[2], gen_lights->mem, gen_lights->mem_size, hipMemcpyHostToDevice);
	hipMemcpy(cuda->mem[3], scene_data, sizeof(t_scene), hipMemcpyHostToDevice);
	hipMemcpy(cuda->mem[4], cameras_data, sizeof(t_cam), hipMemcpyHostToDevice);

	rt_launcher <<< grid_size, threads_per_block >>> ((unsigned int *)cuda->mem[0],
												(char *)cuda->mem[1], gen_objects->mem_size,
												u_time,
												(t_scene *)cuda->mem[3], (t_cam *)cuda->mem[4],
												(char *)cuda->mem[2], gen_lights->mem_size, (int *)cuda->mem[5],
												NULL,
												NULL,
												NULL,
												NULL);
	hipDeviceSynchronize();

	// check for errors
	hipError_t error = hipGetLastError();
	// HANDLE_ERROR(error);
	if (error != hipSuccess)
	{
	 fprintf(stderr, "CUDA1 ERROR: %s \n", hipGetErrorString(error));
	}

	//lecture framebuffer
	HANDLE_ERROR(hipMemcpy(pixel_data, cuda->mem[0], scene_data->win_w * scene_data->win_h * sizeof(int), hipMemcpyDeviceToHost));
	// error = hipMemcpy(pixel_data, cuda->mem[0], scene_data->win_w * scene_data->win_h * sizeof(int), hipMemcpyDeviceToHost);
	// if (error != hipSuccess)
	// {
	//  fprintf(stderr, "CUDA2 ERROR: %s \n", hipGetErrorString(error));
	// }

	//lecture target
	if (scene_data->flag & OPTION_RUN)
	{
		HANDLE_ERROR(hipMemcpy(target, cuda->mem[5], sizeof(int), hipMemcpyDeviceToHost));
		// error = hipMemcpy(target, cuda->mem[5], sizeof(int), hipMemcpyDeviceToHost);
		scene_data->flag ^= OPTION_RUN;
	}
	// if (error != hipSuccess)
	// {
	//  fprintf(stderr, "CUDA3 ERROR: %s \n", hipGetErrorString(error));
	// }
}
