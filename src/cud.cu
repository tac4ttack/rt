#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>
#include <stdbool.h>
#include <stdint.h>

extern "C" {

}


#define BACKCOLOR 0x00999999

#define EPSILON 0.00000000000000000000005
#define EPSILONF 0.0000000005F
#define MAX_DIST 10000000.0
#define SHADOW_BIAS 1000

#define CAM scene->cameras
#define CONES scene->cones
#define CYLIND scene->cylinders
#define LIGHT scene->lights
#define PLANE scene->planes
#define SPHERE scene->spheres
#define ACTIVECAM scene->cameras[scene->active_cam]

#define OPTION_WAVE 				(1 << 1)
#define OPTION_SEPIA				(1 << 2)
#define OPTION_BW					(1 << 3)
#define OPTION_RUN					(1 << 4)
#define OPTION_INVERT				(1 << 7)
#define OPTION_CARTOON_FOUR			(1 << 8)
#define OPTION_STEREO				(1 << 9)
#define OPTION_CARTOON_TWO			(1 << 10)

#define OBJ_FLAG_WAVES				(1 << 1)
#define OBJ_FLAG_CHECKERED			(1 << 2)
#define OBJ_FLAG_DIFF_MAP			(1 << 3)
#define OBJ_FLAG_BUMP_MAP			(1 << 4)
#define OBJ_FLAG_PLANE_LIMIT		(1 << 5)
#define OBJ_FLAG_PLANE_LIMIT_FIX	(1 << 6)

# define OBJ_CAM					1
# define OBJ_LIGHT					2
# define OBJ_CONE					3
# define OBJ_CYLINDER				4
# define OBJ_PLANE					5
# define OBJ_SPHERE					6
# define OBJ_ELLIPSOID				7
# define OBJ_THOR					8
# define OBJ_BOX					9


typedef struct			s_gen
{
	size_t				mem_size;
	size_t				length;
	void				*mem;
	bool				(*add)(struct s_gen *, void *);
	bool				(*remove_mem_index)(struct s_gen *, size_t);
	bool				(*remove_index)(struct s_gen *, size_t);
	void				(*print)(struct s_gen *, \
								void (*)(struct s_gen *, void *));
}						t_gen;

/*
** CAM AND LIGHT STRUCTS ///////////////////////////////////////////////////////
*/

typedef struct			s_float3
{
	float				x;
	float				y;
	float				z;
}						t_float3;
typedef struct			s_cam
{
	unsigned int		id;
	float3				pos;
	float3				dir;
	float				fov;
	float				pitch;
	float				yaw;
	float				roll;
}						t_cam;

typedef struct			s_light
{
	int					size;
	int					type;
	float3				pos;
	float3				dir;
	int					shrink;
	float				brightness;
	int					color;
}						t_light;
////////////////////////////////////////////////////////////////////////////////

/*
** OBJECTS STRUCTURES //////////////////////////////////////////////////////////
*/
typedef struct			s_object
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				dir;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;
}						t_object;

typedef struct			s_box
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				dir;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;

	float3				min;
	float3				max;
}						t_box;

typedef struct			s_cone
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				dir;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;

	float				angle;
	float3				u_axis;
}						t_cone;

typedef struct			s_cylinder
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				dir;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;

	float				height;
	float3				base_dir;
	float				radius;
	float3				u_axis;
}						t_cylinder;

typedef struct			s_plane
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				normal;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;

	float				radius;
	float3				u_axis;
}						t_plane;

typedef struct			s_sphere
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				dir;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;

	float				radius;
}						t_sphere;

typedef struct			s_ellipsoid
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				dir;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;

	float				radius;
	float3				axis_size;
}						t_ellipsoid;

typedef struct			s_thor
{
	int					size;
	int					type;
	int					flags;
	int					id;
	float3				pos;
	float3				dir;
	float3				diff;
	float3				spec;
	int					color;
	float				reflex;
	float				refract;
	float				opacity;
	float3				limit_pos;
	float3				limit_dir;
	float3				waves_p1;
	float3				waves_p2;
	float3				check_size;
	int					diff_map_id;
	float3				diff_offset;
	float3				diff_ratio;
	float3				test_var1;
	float3				test_var2;
	float3				test_var3;

	double				lil_radius;
	double				big_radius;
}						t_thor;
////////////////////////////////////////////////////////////////////////////////

/*
** TREE OF RAYS STRUCT /////////////////////////////////////////////////////////
*/
typedef	struct			s_tor
{
	int					activate;
	float3				pos;
	float3				prim;
	float3				normale;
	float				coef_ref;
	float				coef_tra;
	float				opacity;
	unsigned int		color;
	int				mem_index;
	int					id;
	int					type;
	float				dist;
	float				fr;
	float				ft;
}						t_tor;
////////////////////////////////////////////////////////////////////////////////

/*
** CORE STRUCTS ////////////////////////////////////////////////////////////////
*/
typedef struct			s_light_ray
{
	float3				dir;
	float				dist;
}						t_light_ray;

typedef struct			s_hit
{
	float				dist;
	float3				normal;
	float3				pos;
	t_object 			*obj;
	int					mem_index;
	float				opacity;
	unsigned int		color;
	int					wall; // WIP
}						t_hit;

typedef struct			s_tex
{
	unsigned int		pixel_array[12076155];
	int					width;
	int					height;
}						t_tex;

typedef struct			s_ret
{
	int					wall;
	float				dist;
	float3				normal;
}						t_ret;

typedef struct			s_scene
{
	unsigned int		n_cams;
	unsigned int		active_cam;
	unsigned int		win_w;
	unsigned int		win_h;
	float3				ambient;
	unsigned int		mou_x;
	unsigned int		mou_y;
	unsigned int		depth;
	float				u_time;
	int					flag;
	unsigned int		over_sampling;
	unsigned int		mem_size_obj;
	unsigned int		mem_size_lights;
	float3				check_p1;
	float3				check_p2;
	float3				waves_p1;
	float3				waves_p2;
	t_cam				*cameras;
//	void				*dummy_pedro;
	void				*mem_lights;  //repassé en void à cause de l'erreur compilation, sinon pour oclgrind foutre char
//	void				*dummy_gomez;
	void				*mem_obj; //repassé en void à cause de l'erreur compilation, sinon pour oclgrind foutre char
//	void				*dummy_ramon;
	unsigned int		*texture_earth;
	unsigned int		*texture_moon;
	unsigned int		*texture_earth_cloud;
	unsigned int		*texture_star;
}						t_scene;

__host__ __device__ float dot(const float3 a, const float3 b)
{
	return ((a.x * b.x) + (a.y * b.y) + (a.z * b.z));
}

__host__ __device__ float sqrt_magnitude(const float3 a)
{
	return (sqrt(fabs(a.x * a.x) + fabs(a.y * a.y) + fabs(a.z * a.z)));
}

__host__ __device__ float3 normalize(const float3 a)
{
	float3		newv;
	float		ret_magnitude;

	ret_magnitude = sqrt_magnitude(a);
	if (fabs(ret_magnitude) < EPSILONF)
		return (a);
	newv.x = a.x / ret_magnitude;
	newv.y = a.y / ret_magnitude;
	newv.z = a.z / ret_magnitude;
	return (newv);
}

__host__ __device__ float length(const float3 a)
{
	return (fabs((a.x * a.x)) + fabs((a.y * a.y)) + fabs((a.z * a.z)));
}

inline __host__ __device__ float radians(double degree) {
    return (degree * M_PI / 180.0);
}

// additional constructors
inline __host__ __device__ float3 make_float3(float s)
{
    return make_float3(s, s, s);
}
inline __host__ __device__ float3 make_float3(float2 a)
{
    return make_float3(a.x, a.y, 0.0f);
}
inline __host__ __device__ float3 make_float3(float2 a, float s)
{
    return make_float3(a.x, a.y, s);
}
inline __host__ __device__ float3 make_float3(float4 a)
{
    return make_float3(a.x, a.y, a.z);  // discards w
}
inline __host__ __device__ float3 make_float3(int3 a)
{
    return make_float3(float(a.x), float(a.y), float(a.z));
}

// addition
inline __host__ __device__ float3 operator+(float3 a, float3 b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}
inline __host__ __device__ float3 operator+(float3 a, float b)
{
    return make_float3(a.x + b, a.y + b, a.z + b);
}
inline __host__ __device__ void operator+=(float3 &a, float3 b)
{
    a.x += b.x; a.y += b.y; a.z += b.z;
}

// subtract
inline __host__ __device__ float3 operator-(float3 a, float3 b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}
inline __host__ __device__ float3 operator-(float3 a, float b)
{
    return make_float3(a.x - b, a.y - b, a.z - b);
}
inline __host__ __device__ void operator-=(float3 &a, float3 b)
{
    a.x -= b.x; a.y -= b.y; a.z -= b.z;
}

// multiply
inline __host__ __device__ float3 operator*(float3 a, float3 b)
{
    return make_float3(a.x * b.x, a.y * b.y, a.z * b.z);
}
inline __host__ __device__ float3 operator*(float3 a, float s)
{
    return make_float3(a.x * s, a.y * s, a.z * s);
}
inline __host__ __device__ float3 operator*(float s, float3 a)
{
    return make_float3(a.x * s, a.y * s, a.z * s);
}
inline __host__ __device__ void operator*=(float3 &a, float s)
{
    a.x *= s; a.y *= s; a.z *= s;
}

// divide
inline __host__ __device__ float3 operator/(float3 a, float3 b)
{
    return make_float3(a.x / b.x, a.y / b.y, a.z / b.z);
}
inline __host__ __device__ float3 operator/(float3 a, float s)
{
    float inv = 1.0f / s;
    return a * inv;
}
inline __host__ __device__ float3 operator/(float s, float3 a)
{
    float inv = 1.0f / s;
    return a * inv;
}
inline __host__ __device__ void operator/=(float3 &a, float s)
{
    float inv = 1.0f / s;
    a *= inv;
}

__host__ __device__ t_hit	hit_init(void)
{
	t_hit		hit;

	hit.dist = 0.f;
	hit.normal = make_float3(0.0f);

	hit.obj = NULL; // dangling dangerouss!

	hit.wall = 0;
	hit.color = 0;
	hit.pos = make_float3(0.0f);
	hit.mem_index = 0;
	hit.opacity = 0;
	return (hit);
}

__host__ __device__ float3	rotat_zyx(const float3 vect, const float pitch, const float yaw, const float roll)
{
	float3		res;
	float		rad_pitch = radians(pitch);
	float		rad_yaw = radians(yaw);
	float		rad_roll = radians(roll);

	res.x = vect.x * cos(rad_roll) * cos(rad_yaw) + vect.y * (cos(rad_pitch) * -sin(rad_roll) + cos(rad_roll) * sin(rad_yaw) * sin(rad_pitch)) + vect.z * (-sin(rad_roll) * -sin(rad_pitch) + cos(rad_roll) * sin(rad_yaw) * cos(rad_pitch));
	res.y = vect.x * sin(rad_roll) * cos(rad_yaw) + vect.y * (cos(rad_roll) * cos(rad_pitch) + sin(rad_roll) * sin(rad_yaw) * sin(rad_pitch)) + vect.z * (cos(rad_roll) * -sin(rad_pitch) + sin(rad_roll) * sin(rad_yaw) * cos(rad_pitch));
	res.z = vect.x * -sin(rad_yaw) + vect.y * cos(rad_yaw) * sin(rad_pitch) + vect.z * cos(rad_yaw) * cos(rad_pitch);
	return (res);
}

__host__ __device__ unsigned int	blend_multiply(const unsigned int c1, const unsigned int c2)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);
	unsigned int r2 = (c2 & 0x00FF0000) >> 16;
	unsigned int g2 = (c2 & 0x0000FF00) >> 8;
	unsigned int b2 = (c2 & 0x000000FF);

	r = (r1 * r2 > 255 ? 255 : r1 * r2);
	g = (g1 * g2 > 255 ? 255 : g1 * g2);
	b = (b1 * b2 > 255 ? 255 : b1 * b2);

	return ((r << 16) + (g << 8) + b);
}

__host__ __device__ unsigned int	blend_med(const unsigned int c1, const unsigned int c2)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);
	unsigned int r2 = (c2 & 0x00FF0000) >> 16;
	unsigned int g2 = (c2 & 0x0000FF00) >> 8;
	unsigned int b2 = (c2 & 0x000000FF);

	r =  (r1 + r2) / 2;
	g =  (g1 + g2) / 2;
	b =  (b1 + b2) / 2;
	return ((r << 16) + (g << 8) + b);
}

__host__ __device__ unsigned int	blend_add(const unsigned int c1, const unsigned int c2)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);
	unsigned int r2 = (c2 & 0x00FF0000) >> 16;
	unsigned int g2 = (c2 & 0x0000FF00) >> 8;
	unsigned int b2 = (c2 & 0x000000FF);

	r = (r1 + r2 > 255 ? 255 : r1 + r2);
	g = (g1 + g2 > 255 ? 255 : g1 + g2);
	b = (b1 + b2 > 255 ? 255 : b1 + b2);
	return ((r << 16) + (g << 8) + b);
}

__host__ __device__ unsigned int	blend_factor(const unsigned int c1, const float factor)
{
	unsigned int r, g, b;
	unsigned int r1 = (c1 & 0x00FF0000) >> 16;
	unsigned int g1 = (c1 & 0x0000FF00) >> 8;
	unsigned int b1 = (c1 & 0x000000FF);

	r = r1 * factor;
	g = g1 * factor;
	b = b1 * factor;
	return ((r << 16) + (g << 8) + b);
}

__host__ __device__ unsigned int	get_ambient(const t_scene *scene, const unsigned int obj_color)
{
	unsigned int r, g, b;

	r = (obj_color & 0x00FF0000) >> 16;
	g = (obj_color & 0x0000FF00) >> 8;
	b = (obj_color & 0x000000FF);
	r = (0.01 + r * scene->ambient.x > 255 ? 255 : 0.01 + r * scene->ambient.x);
	g = (0.01 + g * scene->ambient.y > 255 ? 255 : 0.01 + g * scene->ambient.y);
	b = (0.01 + b * scene->ambient.z > 255 ? 255 : 0.01 + b * scene->ambient.z);
	return ((r << 16) + (g << 8) + b);
}

__host__ __device__ unsigned int cartoonize_four(unsigned int col_r, unsigned int col_g, unsigned int col_b)
{
		if (col_r > 0 && col_r <= 50)
			col_r = 32;
		else if (col_r > 50 && col_r <= 100)
			col_r = 96;
		else if (col_r > 100 && col_r <= 200)
			col_r = 160;
		else if (col_r > 200 && col_r < 250)
			col_r = 224;
		else if (col_r >= 255)
			col_r = 255;

		if (col_g > 0 && col_g <= 50)
			col_g = 32;
		else if (col_g > 50 && col_g <= 100)
			col_g = 96;
		else if (col_g > 100 && col_g <= 200)
			col_g = 160;
		else if (col_g > 200 && col_g < 250)
			col_g = 224;
		else if (col_g >= 255)
			col_g = 255;

		if (col_b > 0 && col_b <= 50)
			col_b = 32;
		else if (col_b > 50 && col_b <= 100)
			col_b = 96;
		else if (col_b > 100 && col_b <= 200)
			col_b = 160;
		else if (col_b > 200 && col_b < 250)
			col_b = 224;
		else if (col_b >= 255)
			col_b = 255;

	return (((col_r << 16) + (col_g << 8) + col_b));
}

__host__ __device__ unsigned int cartoonize_two(unsigned int col_r, unsigned int col_g, unsigned int col_b)
{
		if (col_r > 0 && col_r <= 128)
			col_r = 64;
		else if (col_r > 128 && col_r <= 255)
			col_r = 192;
		else if (col_r >= 255)
			col_r = 255;

		if (col_g > 0 && col_g <= 128)
			col_g = 64;
		else if (col_g > 128 && col_g <= 255)
			col_g = 192;
		else if (col_g >= 255)
			col_g = 255;

		if (col_b > 0 && col_r <= 128)
			col_b = 64;
		else if (col_b > 128 && col_b <= 255)
			col_b = 192;
		else if (col_b >= 255)
			col_b = 255;

	return (((col_r << 16) + (col_g << 8) + col_b));
}

__host__ __device__  bool		solve_quadratic(const float a, const float b, const float c, float *inter0, float *inter1)
{
	float 		discr;
	float 		tmp;
	float 		q;

	q = 0;
	discr = b * b - 4 * a * c;
	tmp = 0;
	if (discr < 0)
		return (false);
	else if (discr < EPSILONF)
	{
		*inter0 = -0.5 * b / a;
		*inter1 = *inter0;
	}
	else
	{
		q = (b > 0) ? (-0.5 * (b + sqrt(discr))): (-0.5 * (b - sqrt(discr)));
		*inter0 = q / a;
		*inter1 = c / q;
	}
	if (*inter0 > *inter1)
	{
		tmp = *inter0;
		*inter0 = *inter1;
		*inter1 = tmp;
	}
	if (*inter0 < 0)
	{
		if (*inter1 < 0)
			return (false);
	}
	return (true);
}

__host__ __device__ float3	get_sphere_abc(const float radius, const float3 ray, const float3 origin)
{
	float3		abc = make_float3(0);

	abc.x = dot(ray, ray);
	abc.y = 2 * dot(ray, origin);
	abc.z = dot(origin, origin) - (radius * radius);
	return (abc);
}


__host__ __device__ t_ret	inter_sphere(const t_sphere *sphere, const float3 ray, const float3 origin)
{
	float3		abc = make_float3(0);
	float		res1 = 0;
	float		res2 = 0;
	float3		pos = make_float3(0);
	t_ret		ret;

	ret.dist = 0;
	ret.wall = 0;
	pos = origin - sphere->pos;
	abc = get_sphere_abc(sphere->radius, ray, pos);
	if (!solve_quadratic(abc.x, abc.y, abc.z, &res1, &res2))
		return (ret);
	//if (sphere->flags & OBJ_FLAG_PLANE_LIMIT)
	//	return (object_limited((t_object  *)sphere, res1, res2, ray, origin));
	if ((res1 < res2 && res1 > 0) || (res1 > res2 && res2 < 0))
		ret.dist = res1;
	else
		ret.dist = res2;
	return (ret);

}

__host__ __device__ t_hit			ray_hit(const t_scene *scene, const float3 origin, const float3 ray, float lightdist)
{
	t_hit						hit;
	t_object 					*obj;
	int							mem_index_obj;
	t_ret						ret;

	hit = hit_init();
	mem_index_obj = 0;
	obj = 0;
	if (lightdist == 0)
		hit.opacity = 1;
	while (mem_index_obj < scene->mem_size_obj)
	{
		obj = (t_object *)((char *)scene->mem_obj + mem_index_obj);
		if (obj->type == OBJ_SPHERE)
		 	ret = inter_sphere((struct s_sphere *)obj, ray, origin);
		/*else if (obj->type == OBJ_CYLINDER)
		 	ret = inter_cylinder(( struct s_cylinder *)obj, ray, origin);
		else if (obj->type == OBJ_PLANE)
		 	ret = inter_plan(( struct s_plane *)obj, ray, origin);
		else if (obj->type == OBJ_CONE)
		 	ret = inter_cone(( struct s_cone *)obj, ray, origin);
		else if (obj->type == OBJ_ELLIPSOID)
		   	ret = inter_ellipsoid(( struct s_ellipsoid *)obj, ray, origin);
		else if (obj->type == OBJ_THOR)
			ret = inter_thor(( struct s_thor *)obj, ray, origin);
		if (lightdist > 0 && ret.dist < lightdist && ret.dist > EPSILONF)
			hit.opacity += obj->opacity;*/
		if ((ret.dist < hit.dist || hit.dist == 0) && ret.dist > EPSILONF)
		{
			hit.dist = ret.dist;
			hit.normal = ret.normal;
			hit.wall = ret.wall;
			hit.obj = obj;
			hit.mem_index = mem_index_obj;
/*
			printf("Sphere\nType:%i\nFlags%i\nColor %x\nPos %.2f %.2f %.2f\n\
								Dir %.2f %.2f %.2f\n \
								DIff %.2f %.2f %.2f\n \
								SPec %.2f %.2f %.2f\n \
								",
						obj->type, obj->flags, obj->color,
						obj->pos.x, obj->pos.y, obj->pos.z,
						obj->dir.x, obj->dir.y, obj->dir.z,
						obj->diff.x, obj->diff.y, obj->diff.z,
						obj->spec.x, obj->spec.y, obj->spec.z
						);
						*/
		}
		mem_index_obj += obj->size;
	}
	return (hit);
}

__host__ __device__ float3			get_hit_normal(const t_scene *scene, float3 ray, t_hit hit)
{
	float3				res, save;
	t_object 			*object;

	object = hit.obj;
	res = make_float3(0.f);
	if (hit.wall)
		res = hit.normal;
	else
	{
		if (hit.obj->type == OBJ_SPHERE)
		{
		 	res = hit.pos - hit.obj->pos;
		}
		/*else if (hit.obj->type == OBJ_CYLINDER)
			res = get_cylinder_normal(( t_cylinder *)hit.obj, hit);
		else if (hit.obj->type == OBJ_CONE)
			res = get_cone_normal(( t_cone *)hit.obj, hit);
		else if (hit.obj->type == OBJ_ELLIPSOID)
			res = get_ellipsoid_normal(( t_ellipsoid *)hit.obj, &hit);
		else if (hit.obj->type == OBJ_THOR)
			res = get_thor_normal(( t_thor *)hit.obj, hit.pos);*/
		else if (hit.obj->type == OBJ_PLANE)
		{
			save = ray * -1;
			if (dot(hit.obj->dir, save) < 0)
				res = hit.obj->dir * -1;
			else
				res = hit.obj->dir;
		}
	}
	save = res;
	if (object->flags & OBJ_FLAG_WAVES)
	{
		if (object->type == OBJ_PLANE)
			save.y = res.y + object->waves_p1.x * sin((hit.pos.x + scene->u_time));
		else
		{
			save.x = res.x + object->waves_p1.x * sin(res.y * object->waves_p2.x + scene->u_time);
			save.z = res.z + object->waves_p1.y * sin(res.x * object->waves_p2.y + scene->u_time);
			save.y = res.y + object->waves_p1.z * sin(res.x * object->waves_p2.z + scene->u_time);
		}
	}

	return (normalize(save));
}

__host__ __device__  unsigned int			phong(const  t_scene *scene, const t_hit hit, const float3 ray)
{
	float3					reflect, diffuse, speculos;
	float					tmp, brightness, pow_of_spec;
	unsigned int  			col_r, col_g, col_b, obj_r, obj_g, obj_b, l_r, l_b, l_g, hue_light, hue, res_color;
	int						mem_index_lights, light_color;
	t_object 				*obj;
	t_light 				*light;
	t_light_ray				light_ray;
	t_hit					light_hit;


	tmp = 0;
	reflect = make_float3(0.f);
	speculos = make_float3(0.f);
	diffuse = make_float3(0.f);
	brightness = 0;
	hue_light = 0;
	pow_of_spec = 0;
	light_color = 0;
	mem_index_lights = 0;

	obj = hit.obj;
	/*if ((hit.obj->flags & OBJ_FLAG_CHECKERED) || hit.obj->flags & OBJ_FLAG_DIFF_MAP)
		hue = hit.color;
	else*/
		hue = obj->color;
//	printf("%x\n", hue);
//	return (obj->color);

	col_r = (hue & 0x00FF0000) >> 16;
	col_g = (hue & 0x0000FF00) >> 8;
	col_b = (hue & 0x000000FF);
	col_r = (0.01 + col_r * scene->ambient.x > 255 ? 255 : 0.01 + col_r * scene->ambient.x);
	col_g = (0.01 + col_g * scene->ambient.y > 255 ? 255 : 0.01 + col_g * scene->ambient.y);
	col_b = (0.01 + col_b * scene->ambient.z > 255 ? 255 : 0.01 + col_b * scene->ambient.z);
	res_color = ((col_r << 16) + (col_g << 8) + col_b);

	while (mem_index_lights < scene->mem_size_lights)
	{
		tmp = 0;
		light = (t_light *)((char *)scene->mem_lights + mem_index_lights);
		light_ray.dir = light->pos - hit.pos;
		light_ray.dist = length(light_ray.dir);
		light_ray.dir = normalize(light_ray.dir);
		light_hit = ray_hit(scene, hit.pos, light_ray.dir, light_ray.dist);
		if (!(light_hit.dist < light_ray.dist && light_hit.dist > EPSILON) || (light_hit.opacity < 1 && scene->depth != 0))
		{
			// diffuse part
			tmp = (dot(hit.normal, light_ray.dir));
			if (tmp > EPSILONF)
			{
				brightness = (float )light->brightness;
				diffuse = (float3 )obj->diff;
				hue_light = light->color;
				//printf("%.f %.2f %.2f %.2f %x\n", brightness, diffuse.x, diffuse.y, diffuse.z, hue_light);
				col_r = (res_color & 0xFF0000) >> 16;
				col_g = (res_color & 0x00FF00) >> 8;
				col_b = (res_color & 0x0000FF);
				obj_r = (hue & 0xFF0000) >> 16;
				obj_g = (hue & 0x00FF00) >> 8;
				obj_b = (hue & 0x00000FF);
				//printf("%x = %x | %x | %x\n", hue, col_r, col_g, col_b);
				l_r = (hue_light & 0xFF0000) >> 16;
				l_g = (hue_light & 0x00FF00) >> 8;
				l_b = (hue_light & 0x0000FF);
				col_r += ((l_r * brightness) + obj_r) * tmp * diffuse.x;
				col_g += ((l_g * brightness) + obj_g) * tmp * diffuse.y;
				col_b += ((l_b * brightness) + obj_b) * tmp * diffuse.z;
				//printf("%x\n", hue);
/*
				printf("%x == l_r ([%x] * [%.2f] + %x) * [%.2f] * %.2f\n%x == l_g ([%x] * [%.2f] + %x) * [%.2f] * %.2f\n%x == l_b ([%x] * [%.2f] + %x) * [%.2f] * %.2f\n",
								col_r, l_r, brightness, obj_r, tmp, diffuse.x,
								col_g, l_g, brightness, obj_g, tmp, diffuse.y,
								col_b, l_b, brightness, obj_b, tmp, diffuse.z
							);
*/
				(col_r > 255 ? col_r = 255 : 0);
				// commented lines are failed tonemaping test
			//	col_r = (col_r > 255 ? col_r / (col_r + 1) : col_r);
				(col_g > 255 ? col_g = 255 : 0);
			//	col_g = (col_g > 255 ? col_g / (col_g + 1) : col_g);
				(col_b > 255 ? col_b = 255 : 0);
			//	col_b = (col_b > 255 ? col_b / (col_b + 1) : col_b);

				// if (scene->flag & OPTION_CARTOON_FOUR)
				// 	res_color = cartoonize_four(col_r, col_g, col_b);
				// else if (scene->flag & OPTION_CARTOON_TWO)
				//  	res_color = cartoonize_two(col_r, col_g, col_b);
				// else
				res_color = ((col_r << 16) + (col_g << 8) + col_b);
				printf("%x\n", ((col_r << 16) + (col_g << 8) + col_b));
				return (res_color);
			}
/*
			// specular part
			reflect = normalize(((float)(2.0f * dot(hit.normal, light_ray.dir)) * hit.normal) - light_ray.dir);
			float3 tmp2;
			tmp2.x = -ray.x;
			tmp2.y = -ray.y;
			tmp2.z = -ray.z;
			tmp = dot(reflect, tmp2);
			if (tmp > EPSILONF)
			{
				speculos = obj->spec;
				col_r = (res_color & 0x00FF0000) >> 16;
				col_g = (res_color & 0x0000FF00) >> 8;
				col_b = (res_color & 0x000000FF);

				pow_of_spec = pow(tmp, (light->shrink));
				light_color = light->color;

				col_r += (((light_color & 0xFF0000) >> 16) * pow_of_spec) * speculos.x;
				col_g += ((light_color & 0x00FF00) >> 8) * pow_of_spec * speculos.y;
				col_b += (light_color & 0x0000FF) * pow_of_spec * speculos.z;

				(col_r > 255 ? col_r = 255 : 0);
			//	col_r = (col_r > 255 ? col_r / (col_r + 1) : col_r);
				(col_g > 255 ? col_g = 255 : 0);
			//	col_g = (col_g > 255 ? col_g / (col_g + 1) : col_g);
				(col_b > 255 ? col_b = 255 : 0);
			//	col_b = (col_b > 255 ? col_b / (col_b + 1) : col_b);

				res_color = ((col_r << 16) + (col_g << 8) + col_b);
			}
	*/
	/*		res_color = blend_factor(res_color, ((light_hit.opacity - 1) * -1));
			if (scene->flag & OPTION_CARTOON_FOUR)
			 	res_color = cartoonize_four(col_r, col_g, col_b);
			else if (scene->flag & OPTION_CARTOON_TWO)
			 	res_color = cartoonize_two(col_r, col_g, col_b);
		*/
		}
		mem_index_lights += light->size;
	}
	return (res_color);
}

__host__ __device__ unsigned int	get_pixel_color(const t_scene *scene, float3 ray, int *target, bool isHim)
{
	t_hit			hit;
	int				depth;
	unsigned int	color;
	unsigned int	bounce_color;

	hit = hit_init();
	hit.dist = MAX_DIST;
	depth = scene->depth;
	color = 0x00FF0F0F;
	bounce_color = 0;
	hit = ray_hit(scene, (ACTIVECAM.pos), ray, 0);
	if ((isHim == 1) && (hit.obj != NULL))
		*target = hit.mem_index;
	if (hit.dist > EPSILON && hit.dist < MAX_DIST) // ajout d'une distance max pour virer acnee mais pas fiable a 100%
	{
		hit.pos = (hit.dist * ray) + (ACTIVECAM.pos);
		hit.normal = get_hit_normal(scene, ray, hit);
		hit.pos = hit.pos + (0.001f * hit.normal);
		hit.pos = hit.pos + ((hit.dist / SHADOW_BIAS) * hit.normal);
/*
		if ((hit.obj->type == OBJ_SPHERE) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			hit.color = sphere_texture(normalize(hit.obj->pos - hit.pos), scene->texture_earth, 4915, 2457, (( t_sphere *)hit.obj)->diff_ratio, (( t_sphere *)hit.obj)->diff_offset);
		if ((hit.obj->type == OBJ_SPHERE) && (hit.obj->flags & OBJ_FLAG_CHECKERED))
			hit.color = sphere_checkerboard(normalize(hit.obj->pos - hit.pos), hit.obj->color, hit.obj->check_size);

		if ((hit.obj->type == OBJ_PLANE) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			hit.color = plane_texture(hit.normal, hit.pos, (( t_plane *)hit.obj)->u_axis, (( t_plane *)hit.obj)->diff_ratio, (( t_plane *)hit.obj)->diff_offset, scene->texture_star, 1500, 1500);
		if ((hit.obj->type == OBJ_PLANE) && (hit.obj->flags & OBJ_FLAG_CHECKERED))
			hit.color = plane_checkerboard(hit.normal, hit.pos, hit.obj->color, hit.obj->check_size);

		if ((hit.obj->type == OBJ_CYLINDER) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			hit.color = cylinder_texture(hit.pos - hit.obj->pos, ( t_cylinder *)hit.obj, scene->texture_star, 1500, 1500);

		if ((hit.obj->type == OBJ_CONE) && (hit.obj->flags & OBJ_FLAG_DIFF_MAP))
			hit.color = cone_texture(hit.pos - hit.obj->pos, hit.obj->dir, (( t_cone *)hit.obj)->u_axis, scene->texture_star, 1500, 1500, (( t_cone *)hit.obj)->diff_ratio, (( t_cone *)hit.obj)->diff_offset);
*/
		color = phong(scene, hit, ray);
		//if (((hit.obj->refract != 0 && hit.obj->opacity < 1) || hit.obj->reflex > 0) && depth > 0)
		//	return (fresnel(scene, ray, hit, depth + 1, color));
		//printf("%x\n", color);
		return (color  + 0xFF000000);
		//return (blend_add(color, bounce_color));
	}
	//return (get_ambient(scene, BACKCOLOR));
	return (0xFF0000FF);
}

__host__ __device__ float3		get_ray_cam(t_scene *scene, const int x, const int y, const int width, const int height)
{
	float3			cam_ray = make_float3(0.f);
	float			ratio = ((float)width) / ((float)height);

	cam_ray.x = ((2.f * (((float)x + 0.5f) / (float)width)) - 1.f) * ratio * (tanf(radians(ACTIVECAM.fov / 2.f)));
	cam_ray.y = ((1.f - (2 * (((float)y + 0.5f) / (float)height))) * tanf(radians(ACTIVECAM.fov / 2.f)));
	cam_ray.z = 1.f;
	cam_ray = rotat_zyx(cam_ray, ACTIVECAM.pitch, ACTIVECAM.yaw, 0);
	return(normalize(cam_ray));
}

__host__ __device__ unsigned int	ray_trace(	int				index,
												char			*mem_objects,
												int				mem_size_objects,
												float			u_time,

												t_scene			*scene,
												t_cam			*cameras,

												char			*mem_lights,
												int				mem_size_lights,

												int				*target,
												unsigned int	*texture_earth,
												unsigned int	*texture_moon,
												unsigned int	*texture_earth_cloud,
												unsigned int	*texture_star)
{
	unsigned int	final_color;

	float3			prim_ray;
//	unsigned int	final_color_o[32];
	int				x;
	int				y;

	x = index % scene->win_w;
	y = index / scene->win_w;

	scene->cameras = cameras;
	scene->mem_lights = mem_lights;
	scene->u_time = u_time;
	scene->mem_obj = mem_objects;
	scene->mem_size_obj = mem_size_objects;
	scene->mem_size_lights = mem_size_lights;
	scene->texture_earth = texture_earth;
	scene->texture_earth_cloud = texture_earth_cloud;
	scene->texture_moon = texture_moon;
	scene->texture_star = texture_star;
	if (scene->flag & OPTION_RUN && scene->mou_x + scene->win_h * scene->mou_y == index)
		*target = -1;
	final_color = 0;

	//v2 need oversampling
	prim_ray = get_ray_cam(scene, x, y, scene->win_w, scene->win_h);
	final_color = get_pixel_color(scene, prim_ray, target, (scene->flag & OPTION_RUN && x == scene->mou_x && y == scene->mou_y));

/*	if (scene->flag & OPTION_SEPIA)
		final_color = sepiarize(final_color);
	if (scene->flag & OPTION_BW)
		final_color = desaturate(final_color);
	if (scene->flag & OPTION_INVERT)
		final_color = invert(final_color);*/
/*

	// ALPHA INSERT and RGB SWAP
	int4 swap;
	swap.w = 255;
	swap.x = (final_color & 0x00FF0000) >> 16;
	swap.y = (final_color & 0x0000FF00) >> 8;
	swap.z = (final_color & 0x000000FF);
	final_color = ((swap.w << 24) + (swap.z << 16) + (swap.y << 8) + swap.x);
	((__global unsigned int *)output)[id] = final_color;*/
	return (final_color);
}

__global__ void test(unsigned int *output, unsigned int width, unsigned int height,
							char *mem_objects, int mem_size_objects,
							float u_time,
							t_scene *scene_data, t_cam *cameras_data,
							char *mem_lights, int mem_size_lights)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * width + col;
	output[index] = ray_trace(index, mem_objects, mem_size_objects,
								u_time,
							scene_data, cameras_data,
							mem_lights, mem_size_lights, 0, NULL, NULL, NULL, NULL);
}

extern "C" void render_cuda(unsigned int width, unsigned int height,
							int 		*pixel_data,
							t_gen		*gen_objects,
							t_gen		*gen_lights,
							float		u_time,
							t_scene			*scene_data,
							t_cam			*cameras_data)
{
	dim3					threads_per_block(8, 8);
	dim3					grid_size(width / threads_per_block.x, height / threads_per_block.y);

	/*printf("GPU\n");
	printf("t_cam %zu\n", sizeof(t_cam));
	printf("t_scene %zu\n", sizeof(t_scene));
	printf("t_object %zu\n", sizeof(t_object));
	printf("t_gen %zu\n", sizeof(t_gen));
	printf("t_sphere %zu\n", sizeof(t_sphere));
	printf("t_light %zu\n", sizeof(t_light));
	printf("\n");*/
	//printf("\n{[(%.2f)]}\n\n", cameras_data[0].fov);

	unsigned int *output = NULL;
	hipMalloc(&output, width * height * sizeof(int));
	char *mem_objects = NULL;
	hipMalloc(&mem_objects, gen_objects->mem_size);
	hipMemcpy(mem_objects, gen_objects->mem, gen_objects->mem_size, hipMemcpyHostToDevice);
	char *mem_lights = NULL;
	hipMalloc(&mem_lights, gen_lights->mem_size);
	hipMemcpy(mem_lights, gen_lights->mem, gen_lights->mem_size, hipMemcpyHostToDevice);
	t_scene *scene = NULL;
	hipMalloc(&scene, sizeof(t_scene));
	hipMemcpy(scene, scene_data, sizeof(t_scene), hipMemcpyHostToDevice);
	t_cam *cameras = NULL;
	hipMalloc(&cameras, sizeof(t_cam) * scene_data->n_cams); /////////////////*******************
	hipMemcpy(cameras, cameras_data, sizeof(t_cam), hipMemcpyHostToDevice);

	test <<< grid_size, threads_per_block >>> (output, width, height,
												mem_objects, gen_objects->mem_size,
												u_time,
												scene, cameras,
												mem_lights, gen_lights->mem_size);
	hipDeviceSynchronize();

	// check for errors
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
	  fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
	}
	hipMemcpy(pixel_data, output, width * height * sizeof(int), hipMemcpyDeviceToHost);
	if (output != NULL)
		hipFree(output);
	if (mem_objects != NULL)
		hipFree(mem_objects);
	if (mem_lights != NULL)
		hipFree(mem_lights);
}
